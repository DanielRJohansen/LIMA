#include "hip/hip_runtime.h"
#include "Simulation.cuh"


void Box::moveToDevice() {
	int bytes_total = sizeof(Compound) * n_compounds
		+ sizeof(CompoundState) * MAX_COMPOUNDS * 3
		+ sizeof(NeighborList) * (MAX_SOLVENTS + MAX_COMPOUNDS);
	printf("BOX: moving %.2f MB to device\n", (float)bytes_total * 1e-6);

	compounds = genericMoveToDevice(compounds, MAX_COMPOUNDS);
	bridge_bundle = genericMoveToDevice(bridge_bundle, 1);

	coordarray_circular_queue = genericMoveToDevice(coordarray_circular_queue, MAX_COMPOUNDS * STEPS_PER_LOGTRANSFER);
	solventblockgrid_circular_queue = genericMoveToDevice(solventblockgrid_circular_queue, STEPS_PER_SOLVENTBLOCKTRANSFER);


	compound_neighborlists = genericMoveToDevice(compound_neighborlists, MAX_COMPOUNDS);
	//solvent_neighborlists = genericMoveToDevice(solvent_neighborlists, MAX_SOLVENTS);		// TODO: are we still using these??!

	bonded_particles_lut_manager = genericMoveToDevice(bonded_particles_lut_manager, 1);

	forcefield = genericMoveToDevice(forcefield, 1);

	hipDeviceSynchronize();
	printf("Box transferred to device\n");
}

void Box::deleteMembers(const bool is_on_device) {
	if (is_on_device) {
		hipFree(compounds);
		hipFree(coordarray_circular_queue);
		hipFree(solventblockgrid_circular_queue);
		hipFree(transfermodule_array);

		hipFree(compound_grid);		
		hipFree(compound_neighborlists);

		hipFree(forcefield);

		hipFree(bridge_bundle);
		hipFree(bonded_particles_lut_manager);

		/*hipFree(box->potE_buffer);
		hipFree(box->traj_buffer);
		hipFree(box->outdata);
		hipFree(box->data_GAN);*/
	}
	else {
		delete[] compounds;	// TODO: Finish this
		delete[] bridge_bundle;
		delete[] compound_neighborlists;
		//////delete[] box->solvent_neighborlists;
		delete[] bonded_particles_lut_manager;
	}
}

Box SimUtils::copyToHost(const Box* box_dev) {
	Box box{};
	hipMemcpy(&box, box_dev, sizeof(Box), hipMemcpyDeviceToHost);

	//const Compound** cs = &box.compounds;
	genericCopyToHost(&box.compounds, MAX_COMPOUNDS);
	genericCopyToHost(&box.bridge_bundle, 1);

	genericCopyToHost(&box.coordarray_circular_queue, MAX_COMPOUNDS * STEPS_PER_LOGTRANSFER);
	genericCopyToHost(&box.solventblockgrid_circular_queue, STEPS_PER_SOLVENTBLOCKTRANSFER);

	genericCopyToHost(&box.compound_neighborlists, MAX_COMPOUNDS);
	
	genericCopyToHost(&box.bonded_particles_lut_manager, 1);
	genericCopyToHost(&box.forcefield, 1);


	//genericCopyToHost(box.solvent_neighborlists, MAX_COMPOUNDS);
	//box.compounds = genericCopyToHost(box.compounds, box.n_compounds);
	//box.bridge_bundle = genericCopyToHost(box.bridge_bundle, 1);

	//box.coordarray_circular_queue = genericCopyToHost(box.coordarray_circular_queue)
	printf("Box copied to host\n");
	return box;
}

SimulationDevice::SimulationDevice(const SimParams& params_host, std::unique_ptr<Box> box_host) {
	genericCopyToDevice(params_host, &params, 1);
	
	databuffers = new DatabuffersDevice(box_host->total_particles_upperbound, box_host->n_compounds);
	databuffers = genericMoveToDevice(databuffers, 1);

	box_host->moveToDevice();
	hipMallocManaged(&box, sizeof(Box));
	hipMemcpy(box, box_host.get(), sizeof(Box), hipMemcpyHostToDevice);
	box_host.reset();
}

void SimulationDevice::deleteMembers() {
	box->deleteMembers(true);
	hipFree(box);

	databuffers->freeMembers();
	hipFree(databuffers);

	hipFree(params);
}


Simulation::Simulation(const SimParams& ip) :
	simparams_host{ ip }
{
	box_host = std::make_unique<Box>();
}



Simulation::~Simulation() {
	if (sim_dev != nullptr) {
		sim_dev->deleteMembers();
		hipFree(sim_dev);
	}

}

void Simulation::moveToDevice() {
	if (sim_dev != nullptr) { throw "Expected simdev to be null to move sim to device"; };
	sim_dev = new SimulationDevice(simparams_host, std::move(box_host));
	sim_dev = genericMoveToDevice(sim_dev, 1);
}

void Simulation::copyBoxVariables() {
	n_compounds = box_host->n_compounds;
	n_bridges = box_host->bridge_bundle->n_bridges;


	n_solvents = box_host->n_solvents;
	blocks_per_solventkernel = (int)ceil((float)n_solvents / (float)THREADS_PER_SOLVENTBLOCK);

	compounds_host.resize(n_compounds);
	for (int i = 0; i < n_compounds; i++)
		compounds_host[i] = box_host->compounds[i];

	// Need this variable both on host and device
	total_particles_upperbound = box_host->n_compounds * MAX_COMPOUND_PARTICLES + SolventBlockGrid::blocks_total * MAX_SOLVENTS_IN_BLOCK;
	box_host->total_particles_upperbound = total_particles_upperbound;
}


void InputSimParams::overloadParams(std::map<std::string, double>& dict) {
	overloadParam(dict, &dt, "dt", FEMTO_TO_LIMA);	// convert [fs] to [ls]
	overloadParam(dict, &n_steps, "n_steps");
}

SimParams::SimParams(const InputSimParams& ip) : constparams{ip.n_steps, ip.dt }
{}

DatabuffersDevice::DatabuffersDevice(size_t total_particles_upperbound, int n_compounds) {
	// Permanent Outputs for energy & trajectory analysis
	size_t n_datapoints = total_particles_upperbound * STEPS_PER_LOGTRANSFER;
	printf("Malloc %.2f MB on device for data buffers\n", sizeof(float) * n_datapoints + sizeof(Float3) * n_datapoints);

	hipMallocManaged(&potE_buffer, sizeof(float) * n_datapoints);
	hipMallocManaged(&traj_buffer, sizeof(Float3) * n_datapoints);

#ifdef USEDEBUGF3
	uint64_t bytes_for_debugf3 = sizeof(Float3) * DEBUGDATAF3_NVARS * simulation->total_particles_upperbound * simulation->n_steps;
	hipMallocManaged(&simulation->box->debugdataf3, bytes_for_debugf3);
#endif

	// TRAINING DATA and TEMPRARY OUTPUTS
	int n_loggingdata_device = 10 * STEPS_PER_LOGTRANSFER;
	uint64_t n_traindata_device = static_cast<uint64_t>(N_DATAGAN_VALUES) * MAX_COMPOUND_PARTICLES * n_compounds * STEPS_PER_TRAINDATATRANSFER;
	long double total_bytes = static_cast<long double>(sizeof(float) * static_cast<long double>(n_loggingdata_device) + sizeof(Float3) * n_traindata_device);
	printf("Reserving %.4f MB device mem for logging + training data\n", (float)((total_bytes) * 1e-6));

	hipMallocManaged(&outdata, sizeof(float) * 10 * STEPS_PER_LOGTRANSFER);	// 10 data streams for 10k steps. 1 step at a time.

	hipMallocManaged(&data_GAN, sizeof(Float3) * N_DATAGAN_VALUES * MAX_COMPOUND_PARTICLES * n_compounds * STEPS_PER_TRAINDATATRANSFER);
}

void DatabuffersDevice::freeMembers() {
	hipFree(potE_buffer);
	hipFree(traj_buffer);

	hipFree(outdata);
	hipFree(data_GAN);
}
