#include "hip/hip_runtime.h"
#include "Engine.cuh"

#include "EngineBodies.cuh"
#include "Neighborlists.cuh"
#include "BoundaryCondition.cuh"
#include "SimulationDevice.cuh"
#include "LimaPositionSystem.cuh"
#include "EngineKernels.cuh"
#include "Thermostat.cuh"
#include "SupernaturalForces.cuh"
#include "PME.cuh"

#include "Statistics.h"
#include "Utilities.h"

#include "EngineHostside.h"


Engine::Engine(std::unique_ptr<Simulation> _sim, BoundaryConditionSelect bc, std::unique_ptr<LimaLogger> logger)
	: bc_select(bc), m_logger(std::move(logger)), forceEnergyInterims(_sim->box_host->boxparams.n_compounds, _sim->box_host->boxparams.n_solvents, BoxGrid::BlocksTotal(_sim->box_host->boxparams.boxSize))
{
	simulation = std::move(_sim);

	verifyEngine();

	dataBuffersDevice = std::make_unique<DatabuffersDeviceController>(simulation->box_host->boxparams.total_particles_upperbound, 
		simulation->box_host->boxparams.n_compounds, simulation->simparams_host.data_logging_interval);


	// Create the Sim_dev {
	{
		if (sim_dev != nullptr) { throw std::runtime_error("Expected simdev to be null to move sim to device"); }
		sim_dev = new SimulationDevice(simulation->simparams_host, simulation->box_host.get(), BoxConfig::Create(*simulation->box_host), BoxState::Create(*simulation->box_host), *dataBuffersDevice);
		sim_dev = genericMoveToDevice(sim_dev, 1);
	}
	setDeviceConstantMemory();
	boxStateCopy = std::make_unique<BoxState>(nullptr, nullptr, nullptr, nullptr, nullptr);
	boxConfigCopy = std::make_unique<BoxConfig>(nullptr, nullptr, nullptr, nullptr, nullptr);
	hipMemcpy(boxStateCopy.get(), sim_dev->boxState, sizeof(BoxState), hipMemcpyDeviceToHost);
	hipMemcpy(boxConfigCopy.get(), &sim_dev->boxConfig, sizeof(BoxConfig), hipMemcpyDeviceToHost);
	neighborlistsPtr = sim_dev->compound_neighborlists;
	compoundgridPtr = sim_dev->compound_grid;



    std::vector<ForceField_NB::ParticleParameters> compoundParticleParams(simulation->box_host->boxparams.n_compounds * MAX_COMPOUND_PARTICLES, ForceField_NB::ParticleParameters{0,0});
    for (int cid = 0; cid < simulation->box_host->compounds.size(); cid++) {
        const Compound& compound = simulation->box_host->compounds[cid];
        for (int pid = 0; pid < compound.n_particles; pid++) {
            compoundParticleParams[cid*MAX_COMPOUND_PARTICLES + pid] = simulation->forcefield.particle_parameters[compound.atom_types[pid]];
        }

    }
    compoundLjParameters = GenericCopyToDevice(compoundParticleParams);

	for (hipStream_t& stream : cudaStreams) {
		hipStreamCreate(&stream);
	}
	hipStreamCreate(&pmeStream);

	pmeController = std::make_unique<PME::Controller>(simulation->box_host->boxparams.boxSize, *simulation->box_host, simulation->simparams_host.cutoff_nm, pmeStream);
	hipMalloc(&forceEnergiesPME, sizeof(ForceEnergy) * simulation->box_host->boxparams.n_compounds * MAX_COMPOUND_PARTICLES); // TODO: make hipFree ...
	hipMemset(forceEnergiesPME, 0, sizeof(ForceEnergy) * simulation->box_host->boxparams.n_compounds * MAX_COMPOUND_PARTICLES);

	bondgroups = GenericCopyToDevice(simulation->box_host->bondgroups);
	hipMalloc(&forceEnergiesBondgroups, sizeof(ForceEnergy) * simulation->box_host->bondgroups.size() * BondGroup::maxParticles);






	auto boxparams = simulation->box_host->boxparams;
	thermostat = std::make_unique<Thermostat>(boxparams.n_compounds, boxparams.n_solvents, boxparams.total_particles_upperbound);

	// To create the NLists we need to bootstrap the traj_buffer, since it has no data yet
	bootstrapTrajbufferWithCoords();

	NeighborLists::updateNlists(sim_dev, simulation->getStep(), simulation->simparams_host.bc_select, simulation->box_host->boxparams, timings.nlist);
	m_logger->finishSection("Engine Ready");
}

Engine::~Engine() {
	if (sim_dev != nullptr) {
		sim_dev->FreeMembers();
		hipFree(sim_dev);
	}
	forceEnergyInterims.Free();

	hipFree(compoundLjParameters);
	hipFree(forceEnergiesPME);
	hipFree(forceEnergiesBondgroups);
	hipFree(bondgroups);

	for (hipStream_t& stream : cudaStreams) {
		hipStreamDestroy(stream);
	}

	LIMA_UTILS::genericErrorCheck("Error during Engine destruction");
	assert(simulation == nullptr);
}


void Engine::setDeviceConstantMemory() {
	//const int forcefield_bytes = sizeof(ForceField_NB);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::forcefield), &simulation->forcefield, sizeof(ForceField_NB), 0, hipMemcpyHostToDevice);	// So there should not be a & before the device __constant__
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::tinymolForcefield), &simulation->forcefieldTinymol, sizeof(ForcefieldTinymol), 0, hipMemcpyHostToDevice);

	BoxSize boxSize_host;
	boxSize_host.Set(simulation->box_host->boxparams.boxSize);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::boxSize), &boxSize_host, sizeof(BoxSize), 0, hipMemcpyHostToDevice);
	//SetConstantMem(simulation->boxparams_host.boxSize);
	//BoxSize bs;
	//hipMemcpyFromSymbol(&bs, HIP_SYMBOL(DeviceConstants::boxSize), sizeof(BoxSize));

//	hipDeviceSynchronize();


	//BoxSize bs1;
	//hipMemcpyFromSymbol(&bs1, HIP_SYMBOL(DeviceConstants::boxSize), sizeof(BoxSize));

	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::cutoffNM), &simulation->simparams_host.cutoff_nm, sizeof(float), 0, hipMemcpyHostToDevice);
	const float cutoffNmReciprocal = 1.f / simulation->simparams_host.cutoff_nm;
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::cutoffNmReciprocal), &cutoffNmReciprocal, sizeof(float), 0, hipMemcpyHostToDevice);
	const float cutoffNmSquaredReciprocal = 1.f / (simulation->simparams_host.cutoff_nm * simulation->simparams_host.cutoff_nm );
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::cutoffNmSquaredReciprocal), &cutoffNmSquaredReciprocal, sizeof(float), 0, hipMemcpyHostToDevice);	
	const float ewaldKappa = PhysicsUtils::CalcEwaldkappa(simulation->simparams_host.cutoff_nm);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::ewaldKappa), &ewaldKappa, sizeof(float), 0, hipMemcpyHostToDevice);

	const float initialThermostatScalar = 1.f;
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::thermostatScalar), &initialThermostatScalar, sizeof(float), 0, hipMemcpyHostToDevice);

	assert(simulation->forcefieldTest.size() == ForceField_NB::MAX_TYPES * ForceField_NB::MAX_TYPES);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::nonbondedinteractionParams), simulation->forcefieldTest.data(), sizeof(NonbondedInteractionParams) * simulation->forcefieldTest.size(), 0, hipMemcpyHostToDevice);

	// Prepare precomputed values on device
	const float cutoffNM = simulation->simparams_host.cutoff_nm;
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::bsplineTable), PrecomputeBsplineTable().data(), sizeof(float) * PrecomputeBsplineTable().size(), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::erfcForcescalarTable), PrecomputeErfcForcescalarTable(cutoffNM).data(), sizeof(float) * PrecomputeErfcForcescalarTable(cutoffNM).size(), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::erfcPotentialscalarTable), PrecomputeErfcPotentialscalarTable(cutoffNM).data(), sizeof(float) * PrecomputeErfcPotentialscalarTable(cutoffNM).size(), 0, hipMemcpyHostToDevice);

	LIMA_UTILS::genericErrorCheck("Error while setting CUDA __constant__ memory\n");
}




void Engine::step() {
	LIMA_UTILS::genericErrorCheckNoSync("Error before step!");

	deviceMaster();	// Device first, otherwise offloading data always needs the last datapoint!
	assert(simulation);
	assert(sim_dev);
	simulation->step++;

	hostMaster();

	LIMA_UTILS::genericErrorCheckNoSync("Error after step!");
}

void Engine::hostMaster() {						// This is and MUST ALWAYS be called after the deviceMaster, and AFTER incStep()!
	auto t0 = std::chrono::high_resolution_clock::now();
	if (DatabuffersDeviceController::IsBufferFull(simulation->getStep(), simulation->simparams_host.data_logging_interval)) {
		offloadLoggingData(DatabuffersDeviceController::nStepsInBuffer);
		runstatus.stepForMostRecentData = simulation->getStep();

		if ((simulation->getStep() % simulation->simparams_host.steps_per_temperature_measurement) == 0 && simulation->getStep() > 0) {
			auto [temperature, thermostatScalar] = thermostat->Temperature(sim_dev, simulation->box_host->boxparams, simulation->simparams_host);
			simulation->temperature_buffer.push_back(temperature);
			runstatus.current_temperature = temperature;

			if (simulation->simparams_host.apply_thermostat)
				hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::thermostatScalar), &thermostatScalar, sizeof(float), 0, hipMemcpyHostToDevice);
		}
		
		HandleEarlyStoppingInEM();

		NeighborLists::updateNlists(sim_dev, simulation->getStep(), simulation->simparams_host.bc_select, simulation->box_host->boxparams, timings.nlist);
	}

	// Handle status
	runstatus.current_step = simulation->getStep();
	if (runstatus.current_step >= simulation->simparams_host.n_steps || runstatus.critical_error_occured)
		runstatus.simulation_finished = true;


	const auto t1 = std::chrono::high_resolution_clock::now();
	const int cpu_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
	timings.cpu_master += cpu_duration;
}

void Engine::terminateSimulation() {
	const int64_t stepsReadyToTransfer = DatabuffersDeviceController::StepsReadyToTransfer(simulation->getStep(), simulation->simparams_host.data_logging_interval);
	offloadLoggingData(stepsReadyToTransfer);

	sim_dev->boxState->CopyDataToHost(*simulation->box_host);

	LIMA_UTILS::genericErrorCheck("Error during TerminateSimulation");
}

//--------------------------------------------------------------------------	CPU workload --------------------------------------------------------------//

void Engine::offloadLoggingData(const int64_t steps_to_transfer) {
	assert(steps_to_transfer <= simulation->getStep());
	if (steps_to_transfer == 0) { return; }

	hipDeviceSynchronize();

	const int64_t startstep = simulation->getStep() - steps_to_transfer * simulation->simparams_host.data_logging_interval;
	const int64_t startindex = LIMALOGSYSTEM::getMostRecentDataentryIndex(startstep, simulation->simparams_host.data_logging_interval);
	const int64_t indices_to_transfer = LIMALOGSYSTEM::getNIndicesBetweenSteps(startstep, simulation->getStep(), simulation->simparams_host.data_logging_interval);
	const int particlesUpperbound = simulation->box_host->boxparams.total_particles_upperbound;
	
	hipMemcpyAsync(
		simulation->potE_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->potE_buffer,
		sizeof(float) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);
	
	hipMemcpyAsync(
		simulation->vel_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->vel_buffer,
		sizeof(float) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->forceBuffer->getBufferAtIndex(startindex),
		dataBuffersDevice->forceBuffer,
		sizeof(Float3) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->traj_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->traj_buffer,
		sizeof(Float3) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	step_at_last_traj_transfer = simulation->getStep();
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(LIMALOGSYSTEM::getMostRecentDataentryIndex(simulation->getStep() - 1, simulation->simparams_host.data_logging_interval));
}

void Engine::offloadTrainData() {
#ifdef GENERATETRAINDATA
	uint64_t values_per_step = N_DATAGAN_VALUES * MAX_COMPOUND_PARTICLES * simulation->boxparams_host.n_compounds;
	if (values_per_step == 0) {
		return;	// No data to transfer
	}

	uint64_t step_offset = (simulation->getStep() - STEPS_PER_TRAINDATATRANSFER) * values_per_step;	// fix max_compound to the actual count save LOTS of space!. Might need a file in simout that specifies cnt for loading in other programs...
	hipMemcpy(&simulation->trainingdata[step_offset], dataBuffersDevice->data_GAN, sizeof(Float3) * values_per_step * STEPS_PER_TRAINDATATRANSFER, hipMemcpyDeviceToHost);
	LIMA_UTILS::genericErrorCheck("Cuda error during traindata offloading\n");
#endif
}


void Engine::bootstrapTrajbufferWithCoords() {
	if (simulation->simparams_host.n_steps == 0) return;

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");

	// We need to bootstrap step-0 which is used for traj-buffer
	for (int compound_id = 0; compound_id < simulation->box_host->boxparams.n_compounds; compound_id++) {
		for (int particle_id = 0; particle_id < MAX_COMPOUND_PARTICLES; particle_id++) {
			const Float3 particle_abspos = LIMAPOSITIONSYSTEM::GetAbsolutePositionNM(simulation->box_host->compoundCoordsBuffer[compound_id].origo, simulation->box_host->compoundCoordsBuffer[compound_id].rel_positions[particle_id]);
			simulation->traj_buffer->getCompoundparticleDatapointAtIndex(compound_id, particle_id, 0) = particle_abspos;
		}
	}

	step_at_last_traj_transfer = 0.f;
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(0);

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");
}

void Engine::HandleEarlyStoppingInEM() {
	if (!simulation->simparams_host.em_variant || simulation->getStep() == simulation->simparams_host.n_steps)
		return;
	
	const int minStepsPerCheck = 100;
	if (simulation->getStep() > stepAtLastEarlystopCheck + minStepsPerCheck) {
		const float greatestForce = Statistics::MaxLen(simulation->forceBuffer->GetBufferAtStep(simulation->getStep()-1), simulation->forceBuffer->EntriesPerStep());
		runstatus.greatestForce = greatestForce / KILO; // Convert [J/mol/nm] to [kJ/mol/nm]
		simulation->maxForceBuffer.emplace_back(std::pair<int64_t,float>{ simulation->getStep(), runstatus.greatestForce });

		if (runstatus.greatestForce <= simulation->simparams_host.em_force_tolerance) {
			runstatus.simulation_finished = true;
		}

		stepAtLastEarlystopCheck = simulation->getStep();
	}
}




//--------------------------------------------------------------------------	SIMULATION BEGINS HERE --------------------------------------------------------------//
template <typename BoundaryCondition, bool emvariant, bool computePotE>
void Engine::_deviceMaster() {
	
	const BoxParams& boxparams = simulation->box_host->boxparams;
	const int step = simulation->getStep();
	// #### Initial round of force computations
	hipDeviceSynchronize();

	if (boxparams.n_compounds > 0) {
		compoundFarneighborShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE> 
			<<<boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, cudaStreams[0]>>>
            (*boxStateCopy, *boxConfigCopy, neighborlistsPtr, simulation->simparams_host.enable_electrostatics, 
				forceEnergyInterims.forceEnergyFarneighborShortrange, compoundLjParameters);
		LIMA_UTILS::genericErrorCheckNoSync("Error after compoundFarneighborShortrangeInteractionsKernel");

		compoundImmediateneighborAndSelfShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE> 
			<<<boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, cudaStreams[1] >>> 
			(sim_dev, step, forceEnergyInterims.forceEnergyImmediateneighborShortrange);
		LIMA_UTILS::genericErrorCheckNoSync("Error after compoundImmediateneighborAndSelfShortrangeInteractionsKernel");
	}

	if (boxparams.n_solvents > 0) {
		// Should only use max_compound_particles threads here. and let 1 thread handle multiple solvents
		TinymolCompoundinteractionsKernel<BoundaryCondition, emvariant>
			<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[2]>>>
			(*boxStateCopy, *boxConfigCopy, compoundgridPtr, step, forceEnergyInterims.forceEnergiesCompoundinteractions);
		LIMA_UTILS::genericErrorCheckNoSync("Error after TinymolCompoundinteractionsKernel");

		// TODO: Too many threads, we rarely get close to filling the block
		solventForceKernel<BoundaryCondition, emvariant> 
			<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[3]>>>
			(*boxStateCopy, *boxConfigCopy, step, forceEnergyInterims.forceEnergiesTinymolinteractions);
		LIMA_UTILS::genericErrorCheckNoSync("Error after solventForceKernel");
	}
	
	if (ENABLE_ES_LR && simulation->simparams_host.enable_electrostatics) {
		pmeController->CalcCharges(*boxConfigCopy, *boxStateCopy, boxparams.n_compounds, forceEnergiesPME, pmeStream);
		LIMA_UTILS::genericErrorCheckNoSync("Error after HandleElectrostatics");
	}

	if (simulation->simparams_host.snf_select != None) {
		SnfHandler<BoundaryCondition, emvariant>(cudaStreams[2]);
		LIMA_UTILS::genericErrorCheckNoSync("Error after SupernaturalForces");
	}

	if (!simulation->box_host->bondgroups.empty()) {
		BondgroupsKernel<BoundaryCondition, emvariant> << < simulation->box_host->bondgroups.size(), THREADS_PER_BONDSGROUPSKERNEL, 0, cudaStreams[4]>>> 
			(bondgroups, *boxStateCopy, forceEnergiesBondgroups);
		LIMA_UTILS::genericErrorCheckNoSync("Error after BondgroupsKernel");
	}

	// #### Integration and Transfer kernels
	hipStreamSynchronize(pmeStream);
	for (int i = 0; i < cudaStreams.size(); i++) {
		hipStreamSynchronize(cudaStreams[i]);
	}

	if (boxparams.n_compounds > 0) {
		CompoundIntegrationKernel<BoundaryCondition, emvariant> 
			<<<boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, cudaStreams[0] >> >
			(sim_dev, step, forceEnergyInterims, forceEnergiesBondgroups, forceEnergiesPME);
		LIMA_UTILS::genericErrorCheckNoSync("Error after CompoundIntegrationKernel");
	}

	if (boxparams.n_solvents > 0) {
		const bool isTransferStep = SolventBlocksCircularQueue::isTransferStep(step);
		if (isTransferStep)
			TinymolIntegrationLoggingAndTransferout<BoundaryCondition, emvariant, true> 
				<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[1] >>>
					(sim_dev, step, forceEnergyInterims.forceEnergiesCompoundinteractions, forceEnergyInterims.forceEnergiesTinymolinteractions);
		else
			TinymolIntegrationLoggingAndTransferout<BoundaryCondition, emvariant, false>
				<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[1] >>>
					(sim_dev, step, forceEnergyInterims.forceEnergiesCompoundinteractions, forceEnergyInterims.forceEnergiesTinymolinteractions);
		LIMA_UTILS::genericErrorCheckNoSync("Error after TinymolIntegrationLoggingAndTransferout");

		if (isTransferStep) {
			hipDeviceSynchronize();
			solventTransferKernel<BoundaryCondition> 
				<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlockTransfermodule::max_queue_size, 0, cudaStreams[1]>>> 
				(sim_dev, step);
			LIMA_UTILS::genericErrorCheckNoSync("Error after solventTransferKernel");
		}
	}	
}
template void Engine::_deviceMaster<PeriodicBoundaryCondition, true, true>();
template void Engine::_deviceMaster<PeriodicBoundaryCondition, true, false>();
template void Engine::_deviceMaster<PeriodicBoundaryCondition, false, true>();
template void Engine::_deviceMaster<PeriodicBoundaryCondition, false, false>();
template void Engine::_deviceMaster<NoBoundaryCondition, true, true>();
template void Engine::_deviceMaster<NoBoundaryCondition, true, false>();
template void Engine::_deviceMaster<NoBoundaryCondition, false, true>();
template void Engine::_deviceMaster<NoBoundaryCondition, false, false>();



void Engine::deviceMaster() {

	const bool logData = simulation->getStep() % simulation->simparams_host.data_logging_interval == 0;// TODO maybe log at the final step, not 0th?

	switch (simulation->simparams_host.bc_select) {
	case NoBC:
		if (simulation->simparams_host.em_variant) {
			if (logData) {
				_deviceMaster<NoBoundaryCondition, true, true>();
			}
			else {
				_deviceMaster<NoBoundaryCondition, true, false>();
			}
		}
		else {
			if (logData) {
				_deviceMaster<NoBoundaryCondition, false, true>();
			}
			else {
				_deviceMaster<NoBoundaryCondition, false, false>();
			}
		}
		break;
	case PBC:
		if (simulation->simparams_host.em_variant) {
			if (logData) {
				_deviceMaster<PeriodicBoundaryCondition, true, true>();
			}
			else {
				_deviceMaster<PeriodicBoundaryCondition, true, false>();
			}
		}
		else {
			if (logData) {
				_deviceMaster<PeriodicBoundaryCondition, false, true>();
			}
			else {
				_deviceMaster<PeriodicBoundaryCondition, false, false>();
			}
		}
		break;
	default:
		throw std::runtime_error("Unsupported boundary condition in LAUNCH_GENERIC_KERNEL");
	}
}





// This function must not have changing template or normal arguments for it's kernels, or it will break cudaGraph
template <typename BoundaryCondition, bool emvariant>
void Engine::SnfHandler(hipStream_t& stream) {
	switch (simulation->simparams_host.snf_select) {
	case None:
		break;
	case HorizontalSqueeze:
		SupernaturalForces::ApplyHorizontalSqueeze << < simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> > (sim_dev, simulation->getStep());
		break;
	case HorizontalChargeField:
		CompoundSnfKernel<BoundaryCondition, emvariant>
			<< <simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >>>
			(sim_dev, simulation->box_host->uniformElectricField, forceEnergyInterims.forceEnergyBonds);
		break;
	case BoxEdgePotential:
		if (simulation->box_host->boxparams.n_compounds > 0)
			SupernaturalForces::BoxEdgeForceCompounds << < simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> > (sim_dev, simulation->getStep());
		if (simulation->box_host->boxparams.n_solvents > 0)
			SupernaturalForces::BoxEdgeForceSolvents<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(simulation->box_host->boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, stream>>>(sim_dev, simulation->getStep());
		break;
	}
}