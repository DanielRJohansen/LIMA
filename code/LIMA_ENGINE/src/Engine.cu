#include "hip/hip_runtime.h"
#include "Engine.cuh"

#include "EngineBodies.cuh"
#include "Neighborlists.cuh"
#include "BoundaryCondition.cuh"
#include "SimulationDevice.cuh"
#include "LimaPositionSystem.cuh"
#include "EngineKernels.cuh"
#include "Thermostat.cuh"
#include "SupernaturalForces.cuh"
#include "PME.cuh"

#include "Statistics.h"
#include "Utilities.h"

#include "EngineHostside.h"


Engine::Engine(std::unique_ptr<Simulation> _sim, BoundaryConditionSelect bc, std::unique_ptr<LimaLogger> logger)
	: bc_select(bc), m_logger(std::move(logger)), compoundForceEnergyInterims(_sim->box_host->boxparams.n_compounds)
{
	simulation = std::move(_sim);

	verifyEngine();

	dataBuffersDevice = std::make_unique<DatabuffersDeviceController>(simulation->box_host->boxparams.total_particles_upperbound, 
		simulation->box_host->boxparams.n_compounds, simulation->simparams_host.data_logging_interval);


	// Create the Sim_dev {
	{
		if (sim_dev != nullptr) { throw std::runtime_error("Expected simdev to be null to move sim to device"); }
		sim_dev = new SimulationDevice(simulation->simparams_host, simulation->box_host.get(), BoxConfig::Create(*simulation->box_host), BoxState::Create(*simulation->box_host), *dataBuffersDevice);
		sim_dev = genericMoveToDevice(sim_dev, 1);
	}
	setDeviceConstantMemory();
	boxStateCopy = std::make_unique<BoxState>(nullptr, nullptr, nullptr, nullptr, nullptr);
	boxConfigCopy = std::make_unique<BoxConfig>(nullptr, nullptr, nullptr, nullptr, nullptr);
	hipMemcpy(boxStateCopy.get(), sim_dev->boxState, sizeof(BoxState), hipMemcpyDeviceToHost);
	hipMemcpy(boxConfigCopy.get(), &sim_dev->boxConfig, sizeof(BoxConfig), hipMemcpyDeviceToHost);
	neighborlistsPtr = sim_dev->compound_neighborlists;
	compoundgridPtr = sim_dev->compound_grid;



    std::vector<ForceField_NB::ParticleParameters> compoundParticleParams(simulation->box_host->boxparams.n_compounds * MAX_COMPOUND_PARTICLES, ForceField_NB::ParticleParameters{0,0});
    for (int cid = 0; cid < simulation->box_host->compounds.size(); cid++) {
        const Compound& compound = simulation->box_host->compounds[cid];
        for (int pid = 0; pid < compound.n_particles; pid++) {
            compoundParticleParams[cid*MAX_COMPOUND_PARTICLES + pid] = simulation->forcefield.particle_parameters[compound.atom_types[pid]];
        }

    }
    compoundLjParameters = GenericCopyToDevice(compoundParticleParams);

	hipStreamCreate(&masterStream);
	hipStreamCreate(&pmeStream);

	pmeController = std::make_unique<PME::Controller>(simulation->box_host->boxparams.boxSize, *simulation->box_host, simulation->simparams_host.cutoff_nm, pmeStream);
	hipMalloc(&forceEnergiesPME, sizeof(ForceEnergy) * simulation->box_host->boxparams.n_compounds * MAX_COMPOUND_PARTICLES); // TODO: make hipFree ...
	hipMemset(forceEnergiesPME, 0, sizeof(ForceEnergy) * simulation->box_host->boxparams.n_compounds * MAX_COMPOUND_PARTICLES);

	bondgroups = GenericCopyToDevice(simulation->box_host->bondgroups);
	hipMalloc(&forceEnergiesBondgroups, sizeof(ForceEnergy) * simulation->box_host->bondgroups.size() * BondGroup::maxParticles);






	auto boxparams = simulation->box_host->boxparams;
	thermostat = std::make_unique<Thermostat>(boxparams.n_compounds, boxparams.n_solvents, boxparams.total_particles_upperbound);

	// To create the NLists we need to bootstrap the traj_buffer, since it has no data yet
	bootstrapTrajbufferWithCoords();

	NeighborLists::updateNlists(sim_dev, simulation->getStep(), simulation->simparams_host.bc_select, simulation->box_host->boxparams, timings.nlist);

	// Create kernels graphs. We need for for the steps where we log data, and 1 for all other
	int a = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(step), &a, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(transferOutThisStep), &a, sizeof(bool), 0, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	BuildKernelgraph(kernelsGraph_log, true);
	auto err1 = hipGraphInstantiate(&kernelsGraphExec_log, kernelsGraph_log, nullptr, nullptr, 0);
	LIMA_UTILS::genericErrorCheck("2");
	BuildKernelgraph(kernelsGraph_nolog, false);
	auto err2 = hipGraphInstantiate(&kernelsGraphExec_nolog, kernelsGraph_nolog, nullptr, nullptr, 0);
	LIMA_UTILS::genericErrorCheck("4");


	m_logger->finishSection("Engine Ready");
}

Engine::~Engine() {
	if (sim_dev != nullptr) {
		sim_dev->FreeMembers();
		hipFree(sim_dev);
	}
	compoundForceEnergyInterims.Free();

	hipFree(compoundLjParameters);
	hipFree(forceEnergiesPME);
	hipFree(forceEnergiesBondgroups);
	hipFree(bondgroups);



	LIMA_UTILS::genericErrorCheck("Error during Engine destruction");
	assert(simulation == nullptr);
}


void Engine::setDeviceConstantMemory() {
	//const int forcefield_bytes = sizeof(ForceField_NB);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::forcefield), &simulation->forcefield, sizeof(ForceField_NB), 0, hipMemcpyHostToDevice);	// So there should not be a & before the device __constant__
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::tinymolForcefield), &simulation->forcefieldTinymol, sizeof(ForcefieldTinymol), 0, hipMemcpyHostToDevice);

	BoxSize boxSize_host;
	boxSize_host.Set(simulation->box_host->boxparams.boxSize);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::boxSize), &boxSize_host, sizeof(BoxSize), 0, hipMemcpyHostToDevice);
	//SetConstantMem(simulation->boxparams_host.boxSize);
	//BoxSize bs;
	//hipMemcpyFromSymbol(&bs, HIP_SYMBOL(DeviceConstants::boxSize), sizeof(BoxSize));

//	hipDeviceSynchronize();


	//BoxSize bs1;
	//hipMemcpyFromSymbol(&bs1, HIP_SYMBOL(DeviceConstants::boxSize), sizeof(BoxSize));

	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::cutoffNM), &simulation->simparams_host.cutoff_nm, sizeof(float), 0, hipMemcpyHostToDevice);
	const float cutoffNmReciprocal = 1.f / simulation->simparams_host.cutoff_nm;
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::cutoffNmReciprocal), &cutoffNmReciprocal, sizeof(float), 0, hipMemcpyHostToDevice);
	const float cutoffNmSquaredReciprocal = 1.f / (simulation->simparams_host.cutoff_nm * simulation->simparams_host.cutoff_nm );
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::cutoffNmSquaredReciprocal), &cutoffNmSquaredReciprocal, sizeof(float), 0, hipMemcpyHostToDevice);	
	const float ewaldKappa = PhysicsUtils::CalcEwaldkappa(simulation->simparams_host.cutoff_nm);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::ewaldKappa), &ewaldKappa, sizeof(float), 0, hipMemcpyHostToDevice);

	const float initialThermostatScalar = 1.f;
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::thermostatScalar), &initialThermostatScalar, sizeof(float), 0, hipMemcpyHostToDevice);

	assert(simulation->forcefieldTest.size() == ForceField_NB::MAX_TYPES * ForceField_NB::MAX_TYPES);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::nonbondedinteractionParams), simulation->forcefieldTest.data(), sizeof(NonbondedInteractionParams) * simulation->forcefieldTest.size(), 0, hipMemcpyHostToDevice);

	// Prepare precomputed values on device
	const float cutoffNM = simulation->simparams_host.cutoff_nm;
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::bsplineTable), PrecomputeBsplineTable().data(), sizeof(float) * PrecomputeBsplineTable().size(), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::erfcForcescalarTable), PrecomputeErfcForcescalarTable(cutoffNM).data(), sizeof(float) * PrecomputeErfcForcescalarTable(cutoffNM).size(), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::erfcPotentialscalarTable), PrecomputeErfcPotentialscalarTable(cutoffNM).data(), sizeof(float) * PrecomputeErfcPotentialscalarTable(cutoffNM).size(), 0, hipMemcpyHostToDevice);

	LIMA_UTILS::genericErrorCheck("Error while setting CUDA __constant__ memory\n");
}




void Engine::Step() {
	LIMA_UTILS::genericErrorCheckNoSync("Error before step!");

	deviceMaster();	// Device first, otherwise offloading data always needs the last datapoint!
	assert(simulation);
	assert(sim_dev);
	simulation->step++;

	hostMaster();

	LIMA_UTILS::genericErrorCheckNoSync("Error after step!");
}

void Engine::hostMaster() {						// This is and MUST ALWAYS be called after the deviceMaster, and AFTER incStep()!
	auto t0 = std::chrono::high_resolution_clock::now();
	if (DatabuffersDeviceController::IsBufferFull(simulation->getStep(), simulation->simparams_host.data_logging_interval)) {
		offloadLoggingData(DatabuffersDeviceController::nStepsInBuffer);
		runstatus.stepForMostRecentData = simulation->getStep();

		if ((simulation->getStep() % simulation->simparams_host.steps_per_temperature_measurement) == 0 && simulation->getStep() > 0) {
			auto [temperature, thermostatScalar] = thermostat->Temperature(sim_dev, simulation->box_host->boxparams, simulation->simparams_host);
			simulation->temperature_buffer.push_back(temperature);
			runstatus.current_temperature = temperature;

			if (simulation->simparams_host.apply_thermostat)
				hipMemcpyToSymbol(HIP_SYMBOL(DeviceConstants::thermostatScalar), &thermostatScalar, sizeof(float), 0, hipMemcpyHostToDevice);
		}
		
		HandleEarlyStoppingInEM();

		NeighborLists::updateNlists(sim_dev, simulation->getStep(), simulation->simparams_host.bc_select, simulation->box_host->boxparams, timings.nlist);
	}

	// Handle status
	runstatus.current_step = simulation->getStep();
	if (runstatus.current_step >= simulation->simparams_host.n_steps || runstatus.critical_error_occured)
		runstatus.simulation_finished = true;


	const auto t1 = std::chrono::high_resolution_clock::now();
	const int cpu_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
	timings.cpu_master += cpu_duration;
}

void Engine::terminateSimulation() {
	const int64_t stepsReadyToTransfer = DatabuffersDeviceController::StepsReadyToTransfer(simulation->getStep(), simulation->simparams_host.data_logging_interval);
	offloadLoggingData(stepsReadyToTransfer);

	sim_dev->boxState->CopyDataToHost(*simulation->box_host);

	LIMA_UTILS::genericErrorCheck("Error during TerminateSimulation");
}

//--------------------------------------------------------------------------	CPU workload --------------------------------------------------------------//

void Engine::offloadLoggingData(const int64_t steps_to_transfer) {
	assert(steps_to_transfer <= simulation->getStep());
	if (steps_to_transfer == 0) { return; }

	hipDeviceSynchronize();

	const int64_t startstep = simulation->getStep() - steps_to_transfer * simulation->simparams_host.data_logging_interval;
	const int64_t startindex = LIMALOGSYSTEM::getMostRecentDataentryIndex(startstep, simulation->simparams_host.data_logging_interval);
	const int64_t indices_to_transfer = LIMALOGSYSTEM::getNIndicesBetweenSteps(startstep, simulation->getStep(), simulation->simparams_host.data_logging_interval);
	const int particlesUpperbound = simulation->box_host->boxparams.total_particles_upperbound;
	
	hipMemcpyAsync(
		simulation->potE_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->potE_buffer,
		sizeof(float) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);
	
	hipMemcpyAsync(
		simulation->vel_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->vel_buffer,
		sizeof(float) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->forceBuffer->getBufferAtIndex(startindex),
		dataBuffersDevice->forceBuffer,
		sizeof(Float3) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->traj_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->traj_buffer,
		sizeof(Float3) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	step_at_last_traj_transfer = simulation->getStep();
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(LIMALOGSYSTEM::getMostRecentDataentryIndex(simulation->getStep() - 1, simulation->simparams_host.data_logging_interval));
}

void Engine::offloadTrainData() {
#ifdef GENERATETRAINDATA
	uint64_t values_per_step = N_DATAGAN_VALUES * MAX_COMPOUND_PARTICLES * simulation->boxparams_host.n_compounds;
	if (values_per_step == 0) {
		return;	// No data to transfer
	}

	uint64_t step_offset = (simulation->getStep() - STEPS_PER_TRAINDATATRANSFER) * values_per_step;	// fix max_compound to the actual count save LOTS of space!. Might need a file in simout that specifies cnt for loading in other programs...
	hipMemcpy(&simulation->trainingdata[step_offset], dataBuffersDevice->data_GAN, sizeof(Float3) * values_per_step * STEPS_PER_TRAINDATATRANSFER, hipMemcpyDeviceToHost);
	LIMA_UTILS::genericErrorCheck("Cuda error during traindata offloading\n");
#endif
}


void Engine::bootstrapTrajbufferWithCoords() {
	if (simulation->simparams_host.n_steps == 0) return;

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");

	// We need to bootstrap step-0 which is used for traj-buffer
	for (int compound_id = 0; compound_id < simulation->box_host->boxparams.n_compounds; compound_id++) {
		for (int particle_id = 0; particle_id < MAX_COMPOUND_PARTICLES; particle_id++) {
			const Float3 particle_abspos = LIMAPOSITIONSYSTEM::GetAbsolutePositionNM(simulation->box_host->compoundCoordsBuffer[compound_id].origo, simulation->box_host->compoundCoordsBuffer[compound_id].rel_positions[particle_id]);
			simulation->traj_buffer->getCompoundparticleDatapointAtIndex(compound_id, particle_id, 0) = particle_abspos;
		}
	}

	step_at_last_traj_transfer = 0.f;
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(0);

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");
}

void Engine::HandleEarlyStoppingInEM() {
	if (!simulation->simparams_host.em_variant || simulation->getStep() == simulation->simparams_host.n_steps)
		return;
	
	const int minStepsPerCheck = 100;
	if (simulation->getStep() > stepAtLastEarlystopCheck + minStepsPerCheck) {
		const float greatestForce = Statistics::MaxLen(simulation->forceBuffer->GetBufferAtStep(simulation->getStep()-1), simulation->forceBuffer->EntriesPerStep());
		runstatus.greatestForce = greatestForce / KILO; // Convert [J/mol/nm] to [kJ/mol/nm]
		simulation->maxForceBuffer.emplace_back(std::pair<int64_t,float>{ simulation->getStep(), runstatus.greatestForce });

		if (runstatus.greatestForce <= simulation->simparams_host.em_force_tolerance) {
			runstatus.simulation_finished = true;
		}

		stepAtLastEarlystopCheck = simulation->getStep();
	}
}




//--------------------------------------------------------------------------	SIMULATION BEGINS HERE --------------------------------------------------------------//


void Engine::deviceMaster() {
	const BoxParams& boxparams = simulation->box_host->boxparams;
	//const int step = simulation->getStep();
	// #### Initial round of force computations
	hipDeviceSynchronize();

	const bool logData = simulation->getStep() % simulation->simparams_host.data_logging_interval == 0;// TODO maybe log at the final step, not 0th?

	// Overwrite the "step" argument for all kernels, somehow????
	const int _step = simulation->getStep();
	const bool isTransferStep = SolventBlocksCircularQueue::isTransferStep(step);

	hipMemcpyToSymbol(HIP_SYMBOL(step), &_step, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(transferOutThisStep), &isTransferStep, sizeof(bool), 0, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipError_t err;

	if (logData)
		err = hipGraphLaunch(kernelsGraphExec_log, masterStream);
	else
		err = hipGraphLaunch(kernelsGraphExec_nolog, masterStream);

	if (ENABLE_ES_LR && simulation->simparams_host.enable_electrostatics) {
		pmeController->CalcCharges(*boxConfigCopy, *boxStateCopy, boxparams.n_compounds, forceEnergiesPME, pmeStream);
	}

	hipStreamSynchronize(masterStream);
	hipStreamSynchronize(pmeStream);


	if (boxparams.n_compounds > 0) {
		CompoundIntegrationKernel<PeriodicBoundaryCondition, false>
			<< <boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, masterStream >> >
			(sim_dev, compoundForceEnergyInterims, forceEnergiesBondgroups, forceEnergiesPME);
	}

	if (boxparams.n_solvents > 0) {
		TinymolIntegrationLoggingAndTransferout<PeriodicBoundaryCondition, false>
			<< <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, pmeStream >> >
			(sim_dev);
		solventTransferKernel<PeriodicBoundaryCondition>
			<< <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlockTransfermodule::max_queue_size, 0, pmeStream >> >
			(sim_dev);
	}

	hipStreamSynchronize(masterStream);
	hipStreamSynchronize(pmeStream);


	/*hipDeviceSynchronize();
	LIMA_UTILS::genericErrorCheckNoSync("Error after step!");*/

}

template <typename BoundaryCondition, bool emvariant, bool computePotE>
void Engine::_BuildKernelgraph(hipGraph_t& graph) {

	const BoxParams boxparams = simulation->box_host->boxparams;

	hipGraphCreate(&graph, 0);

	hipGraphNode_t nodeFarneighbor = nullptr;
	hipGraphNode_t nodeBonds = nullptr;
	hipGraphNode_t nodeTinymolCompoundinteractions = nullptr;
	hipGraphNode_t nodeTinymolForceKernel = nullptr;
	hipGraphNode_t nodeSnf = nullptr;

	hipGraphNode_t nodeBarrierSection = nullptr;

	hipGraphNode_t nodeCompoundIntegration = nullptr;
	hipGraphNode_t nodeTinymolIntegration = nullptr;
	hipGraphNode_t nodeSolventTransfer = nullptr;

	hipGraphNode_t pmeNode1;

	std::vector<hipGraphNode_t> forceDependencies; // Add all nodes to this vector, so they are sync before moving to integration


	if (boxparams.n_compounds > 0) {
		{
			void* kernelArgs[] = {
				(void*)&(*boxStateCopy),
				(void*)&(*boxConfigCopy),
				(void*)&neighborlistsPtr,
				(void*)&simulation->simparams_host.enable_electrostatics,
				(void*)&compoundForceEnergyInterims.forceEnergyFarneighborShortrange,
				(void*)&compoundLjParameters
			};
			hipKernelNodeParams kernelnodeParams{
				(void*)compoundFarneighborShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE>,
				dim3(boxparams.n_compounds),
				dim3(MAX_COMPOUND_PARTICLES),
				0,
				kernelArgs,
				nullptr
			};

			LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&nodeFarneighbor, graph, nullptr, 0, &kernelnodeParams));
			forceDependencies.emplace_back(nodeFarneighbor);
		}

		{
			hipGraphNode_t node;
			void* kernelArgs[] = { &sim_dev, &compoundForceEnergyInterims.forceEnergyImmediateneighborShortrange };
			hipKernelNodeParams kernelnodeParams = {
				(void*)compoundImmediateneighborAndSelfShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE>,
				boxparams.n_compounds,
				MAX_COMPOUND_PARTICLES,
				0,
				kernelArgs,
				nullptr 
			};
						
			LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&node, graph, nullptr, 0, &kernelnodeParams));
			forceDependencies.emplace_back(node);
		}
	}


	if (!simulation->box_host->bondgroups.empty()) {
		void* kernelArgs[] = { &bondgroups, &(*boxStateCopy), &forceEnergiesBondgroups };
		hipKernelNodeParams kernelnodeParams = {
			(void*)BondgroupsKernel<BoundaryCondition, emvariant>,
			simulation->box_host->bondgroups.size(),
			THREADS_PER_BONDSGROUPSKERNEL,
			0,
			kernelArgs,
			nullptr
		};
		LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&nodeBonds, graph, nullptr, 0, &kernelnodeParams));
		forceDependencies.emplace_back(nodeBonds);
	}

	if (boxparams.n_solvents > 0) {
		{
			void* kernelArgs[] = {
				(void*)&(*boxStateCopy),
				(void*)&(*boxConfigCopy),
				&compoundgridPtr
			};
			hipKernelNodeParams kernelnodeParams = {
				(void*)TinymolCompoundinteractionsKernel<BoundaryCondition, emvariant>,
				BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)),
				SolventBlock::MAX_SOLVENTS_IN_BLOCK,
				0,
				kernelArgs,
				nullptr
			};
			LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&nodeTinymolCompoundinteractions, graph, nullptr, 0, &kernelnodeParams));
			forceDependencies.emplace_back(nodeTinymolCompoundinteractions);
		}
		{
			void* kernelArgs[] = { (void*)&(*boxStateCopy), (void*)&(*boxConfigCopy) };
			hipKernelNodeParams kernelnodeParams = {
				(void*)solventForceKernel<BoundaryCondition, emvariant>,
				BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)),
				SolventBlock::MAX_SOLVENTS_IN_BLOCK,
				0,
				kernelArgs,
				nullptr
			};
			LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&nodeTinymolForceKernel, graph, nullptr, 0, &kernelnodeParams));
			forceDependencies.emplace_back(nodeTinymolForceKernel);
		}
	}
	//if (ENABLE_ES_LR && simulation->simparams_host.enable_electrostatics) {
	//	pmeController->AddToGraph(*boxConfigCopy, *boxStateCopy, boxparams.n_compounds, forceEnergiesPME, graph, forceDependencies, pmeNode1);
	//}
	switch (simulation->simparams_host.snf_select)
	{

	case None:
		break;
	case HorizontalSqueeze:	{
		//SupernaturalForces::ApplyHorizontalSqueeze << < simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> > (sim_dev, simulation->getStep());
		throw std::runtime_error("Not implemented");

		//void* kernelArgs[] = { &sim_dev, &compoundForceEnergyInterims.forceEnergyBonds };
		//hipKernelNodeParams kernelnodeParams = {
		//	(void*)SupernaturalForces::ApplyHorizontalSqueeze
		//	boxparams.n_compounds,
		//	MAX_COMPOUND_PARTICLES,
		//	0,
		//	kernelArgs,
		//	nullptr
		//};
		//LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&nodeSnf, graph, nullptr, 0, &kernelnodeParams));
		//forceDependencies.emplace_back(nodeSnf);
		break;
	}		
	case HorizontalChargeField: {
		void* kernelArgs[] = { &sim_dev, &simulation->box_host->uniformElectricField, &compoundForceEnergyInterims.forceEnergyBonds };
		hipKernelNodeParams kernelnodeParams = {
			(void*)CompoundSnfKernel<BoundaryCondition, emvariant>,
			boxparams.n_compounds,
			MAX_COMPOUND_PARTICLES,
			0,
			kernelArgs,
			nullptr
		};
		LIMA_UTILS::genericErrorCheck(hipGraphAddKernelNode(&nodeSnf, graph, nullptr, 0, &kernelnodeParams));
		forceDependencies.emplace_back(nodeSnf);
		break;

		//CompoundSnfKernel<BoundaryCondition, emvariant>
		//	<< <simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> >
		//	(sim_dev, simulation->box_host->uniformElectricField, compoundForceEnergyInterims.forceEnergyBonds);
	}

	case BoxEdgePotential: {
		throw std::runtime_error("Not implemented");
		/*if (simulation->box_host->boxparams.n_compounds > 0)
			SupernaturalForces::BoxEdgeForceCompounds << < simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> > (sim_dev, simulation->getStep());
		if (simulation->box_host->boxparams.n_solvents > 0)
			SupernaturalForces::BoxEdgeForceSolvents << <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(simulation->box_host->boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, stream >> > (sim_dev, simulation->getStep());
		break;*/
		break;
	}
		
	default:
		break;
	}













	// Sync all prev nodes
	{
		auto err = hipGraphAddEmptyNode(&nodeBarrierSection, graph, forceDependencies.data(), forceDependencies.size());
		LIMA_UTILS::genericErrorCheck("e");
	}

	//if (boxparams.n_compounds > 0) {
	//	void* compoundIntegrationArgs[] = {
	//		&sim_dev,
	//		&compoundForceEnergyInterims,
	//		&forceEnergiesBondgroups,
	//		&forceEnergiesPME
	//	};
	//	hipKernelNodeParams compoundIntegrationParams = {
	//		(void*)CompoundIntegrationKernel<BoundaryCondition, emvariant>,
	//		dim3(boxparams.n_compounds),
	//		dim3(MAX_COMPOUND_PARTICLES),
	//		0,
	//		compoundIntegrationArgs,
	//		nullptr
	//	};
	//	hipGraphAddKernelNode(&nodeCompoundIntegration, graph, &nodeBarrierSection, 1, &compoundIntegrationParams);
	//	LIMA_UTILS::genericErrorCheck("e");
	//}

	//if (boxparams.n_solvents > 0) {
	//	{
	//		void* tinymolIntegrationArgs[] = { &sim_dev };
	//		hipKernelNodeParams tinymolIntegrationParams = {
	//			(void*)TinymolIntegrationLoggingAndTransferout<BoundaryCondition, emvariant>,
	//			dim3(BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize))),
	//			dim3(SolventBlock::MAX_SOLVENTS_IN_BLOCK),
	//			0,
	//			tinymolIntegrationArgs,
	//			nullptr
	//		};
	//		hipGraphAddKernelNode(&nodeTinymolIntegration, graph, &nodeBarrierSection, 1, &tinymolIntegrationParams);
	//		LIMA_UTILS::genericErrorCheck("e");
	//	}
	//	{
	//		void* solventTransferArgs[] = { &sim_dev };
	//		hipKernelNodeParams solventTransferParams = {
	//			(void*)solventTransferKernel<BoundaryCondition>,
	//			dim3(BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize))),
	//			dim3(SolventBlockTransfermodule::max_queue_size),
	//			0,
	//			solventTransferArgs,
	//			nullptr
	//		};
	//		hipGraphAddKernelNode(&nodeSolventTransfer, graph, &nodeTinymolIntegration, 1, &solventTransferParams);
	//		LIMA_UTILS::genericErrorCheck("e");
	//	}
	//}
}
template void Engine::_BuildKernelgraph<PeriodicBoundaryCondition, true, true>(hipGraph_t&);
template void Engine::_BuildKernelgraph<PeriodicBoundaryCondition, true, false>(hipGraph_t&);
template void Engine::_BuildKernelgraph<PeriodicBoundaryCondition, false, true>(hipGraph_t&);
template void Engine::_BuildKernelgraph<PeriodicBoundaryCondition, false, false>(hipGraph_t&);
template void Engine::_BuildKernelgraph<NoBoundaryCondition, true, true>(hipGraph_t&);
template void Engine::_BuildKernelgraph<NoBoundaryCondition, true, false>(hipGraph_t&);
template void Engine::_BuildKernelgraph<NoBoundaryCondition, false, true>(hipGraph_t&);
template void Engine::_BuildKernelgraph<NoBoundaryCondition, false, false>(hipGraph_t&);



void Engine::BuildKernelgraph(hipGraph_t& graph, bool logData) {

	switch (simulation->simparams_host.bc_select) {
	case NoBC:
		if (simulation->simparams_host.em_variant) {
			if (logData) {
				_BuildKernelgraph<NoBoundaryCondition, true, true>(graph);
			}
			else {
				_BuildKernelgraph<NoBoundaryCondition, true, false>(graph);
			}
		}
		else {
			if (logData) {
				_BuildKernelgraph<NoBoundaryCondition, false, true>(graph);
			}
			else {
				_BuildKernelgraph<NoBoundaryCondition, false, false>(graph);
			}
		}
		break;
	case PBC:
		if (simulation->simparams_host.em_variant) {
			if (logData) {
				_BuildKernelgraph<PeriodicBoundaryCondition, true, true>(graph);
			}
			else {
				_BuildKernelgraph<PeriodicBoundaryCondition, true, false>(graph);
			}
		}
		else {
			if (logData) {
				_BuildKernelgraph<PeriodicBoundaryCondition, false, true>(graph);
			}
			else {
				_BuildKernelgraph<PeriodicBoundaryCondition, false, false>(graph);
			}
		}
		break;
	default:
		throw std::runtime_error("Unsupported boundary condition in LAUNCH_GENERIC_KERNEL");
	}
}





// This function must not have changing template or normal arguments for it's kernels, or it will break cudaGraph
template <typename BoundaryCondition, bool emvariant>
void Engine::SnfHandler(hipStream_t& stream) {
	switch (simulation->simparams_host.snf_select) {
	case None:
		break;
	case HorizontalSqueeze:
		SupernaturalForces::ApplyHorizontalSqueeze << < simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> > (sim_dev, simulation->getStep());
		break;
	case HorizontalChargeField:
		CompoundSnfKernel<BoundaryCondition, emvariant>
			<< <simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >>>
			(sim_dev, simulation->box_host->uniformElectricField, compoundForceEnergyInterims.forceEnergyBonds);
		break;
	case BoxEdgePotential:
		if (simulation->box_host->boxparams.n_compounds > 0)
			SupernaturalForces::BoxEdgeForceCompounds << < simulation->box_host->boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, stream >> > (sim_dev, simulation->getStep());
		if (simulation->box_host->boxparams.n_solvents > 0)
			SupernaturalForces::BoxEdgeForceSolvents<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(simulation->box_host->boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, stream>>>(sim_dev, simulation->getStep());
		break;
	}
}





//hipKernelNodeParams kFarParams{};
//// For demonstration, we show typical fields:
//kFarParams.func = (void*)compoundFarneighborShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE>;
//kFarParams.gridDim = dim3(boxparams.n_compounds);
//kFarParams.blockDim = dim3(MAX_COMPOUND_PARTICLES);
//kFarParams.sharedMemBytes = 0;
//{
//	// Put real arguments in a vector<void*>
//	// Adjust these to match your kernel signature
//	static bool cpe = true; // or false, per your usage
//	void* args[] = {
//		&boxStateCopy,
//		&boxConfigCopy,
//		&neighborlistsPtr,
//		&compoundForceEnergyInterims.forceEnergyFarneighborShortrange,
//		&compoundLjParameters
//	};
//	kFarParams.kernelParams = args;
//}
//kFarParams.extra = nullptr;

//// No dependencies  can be added with a null dependency array
//auto err = hipGraphAddKernelNode(&nodeFarneighbor, graph, nullptr, 0, &kFarParams);







// ------------------------------------------------------------------
// 3) Add a barrier node so that Section 2 will start only after
//    both kernels in Section 1 have completed.
// ------------------------------------------------------------------
   //{
   //	hipGraphNode_t dependencies[2] = { nodeFarneighbor, nodeImmediateneighbor };
   //	hipGraphAddEmptyNode(&nodeBarrierSection1, graph, dependencies, 2);
   //}





   // ------------------------------------------------------------------
  // 4) Section 2
  //    - CompoundIntegrationKernel (if n_compounds > 0)
  //    - TinymolIntegrationLoggingAndTransferout + solventTransferKernel (in sequence)
  // ------------------------------------------------------------------
//if (boxparams.n_compounds > 0)
//{
	//hipKernelNodeParams compoundIntegrationParams = {};
	//compoundIntegrationParams.func = (void*)CompoundIntegrationKernel<BoundaryCondition, emvariant>;
	//compoundIntegrationParams.gridDim = dim3(boxparams.n_compounds);
	//compoundIntegrationParams.blockDim = dim3(MAX_COMPOUND_PARTICLES);
	//compoundIntegrationParams.sharedMemBytes = 0;
	//void* compoundIntegrationArgs[] = {
	//	&sim_dev,
	//	&compoundForceEnergyInterims,
	//	&forceEnergiesBondgroups,
	//	&forceEnergiesPME
	//};
	//compoundIntegrationParams.kernelParams = compoundIntegrationArgs;
	//compoundIntegrationParams.extra = nullptr;

	//hipGraphAddKernelNode(&nodeCompoundIntegration, graph, &nodeBarrierSection1, 1, &compoundIntegrationParams);
//}
//
//// ------------------------------------------------------------------
//// 4b) If n_solvents > 0, run 2 kernels in sequence
//// ------------------------------------------------------------------
//if (boxparams.n_solvents > 0)
//{
//	//hipKernelNodeParams tinymolIntegrationParams = {};
//	//tinymolIntegrationParams.func = (void*)TinymolIntegrationLoggingAndTransferout<BoundaryCondition, emvariant>;
//	//tinymolIntegrationParams.gridDim = dim3(BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)));
//	//tinymolIntegrationParams.blockDim = dim3(SolventBlock::MAX_SOLVENTS_IN_BLOCK);
//	//tinymolIntegrationParams.sharedMemBytes = 0;
//	//void* tinymolIntegrationArgs[] = { &sim_dev };
//	//tinymolIntegrationParams.kernelParams = tinymolIntegrationArgs;
//	//tinymolIntegrationParams.extra = nullptr;
//
//	//hipGraphAddKernelNode(&nodeTinymolIntegration, graph, &nodeBarrierSection1, 1, &tinymolIntegrationParams);
//
//	//hipKernelNodeParams solventTransferParams = {};
//	//solventTransferParams.func = (void*)solventTransferKernel<BoundaryCondition>;
//	//solventTransferParams.gridDim = dim3(BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)));
//	//solventTransferParams.blockDim = dim3(SolventBlockTransfermodule::max_queue_size);
//	//solventTransferParams.sharedMemBytes = 0;
//	//void* solventTransferArgs[] = { &sim_dev };
//	//solventTransferParams.kernelParams = solventTransferArgs;
//	//solventTransferParams.extra = nullptr;
//
//	//hipGraphAddKernelNode(&nodeSolventTransfer, graph, &nodeTinymolIntegration, 1, &solventTransferParams);
//}

// #### Force kernels

//if (boxparams.n_compounds > 0) {
//	compoundFarneighborShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE> 
//		<<<boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, cudaStreams[0] >> >
//           (*boxStateCopy, *boxConfigCopy, neighborlistsPtr, simulation->simparams_host.enable_electrostatics, 
   //			compoundForceEnergyInterims.forceEnergyFarneighborShortrange, compoundLjParameters);

   //	/*compoundImmediateneighborAndSelfShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE> 
   //		<<<boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, cudaStreams[1] >>> 
   //		(sim_dev, compoundForceEnergyInterims.forceEnergyImmediateneighborShortrange);*/
   //}

   //if (boxparams.n_solvents > 0) {
   //	// Should only use max_compound_particles threads here. and let 1 thread handle multiple solvents
   //	TinymolCompoundinteractionsKernel<BoundaryCondition, emvariant>
   //		<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[2]>>>
   //		(*boxStateCopy, *boxConfigCopy, compoundgridPtr);

   //	// TODO: Too many threads, we rarely get close to filling the block
   //	solventForceKernel<BoundaryCondition, emvariant> 
   //		<<<BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[3]>>>
   //		(*boxStateCopy, *boxConfigCopy);
   //}

   //if (ENABLE_ES_LR && simulation->simparams_host.enable_electrostatics) {
   //	pmeController->CalcCharges(*boxConfigCopy, *boxStateCopy, boxparams.n_compounds, forceEnergiesPME, pmeStream);
   //}

   /*if (simulation->simparams_host.snf_select != None) {
	   SnfHandler<BoundaryCondition, emvariant>(cudaStreams[2]);
   }*/

   //if (!simulation->box_host->bondgroups.empty()) {
   //	BondgroupsKernel<BoundaryCondition, emvariant> << < simulation->box_host->bondgroups.size(), THREADS_PER_BONDSGROUPSKERNEL, 0, cudaStreams[4]>>> 
   //		(bondgroups, *boxStateCopy, forceEnergiesBondgroups);
   //}

   //// #### Sync all streams
   //for (int i = 0; i < 5; i++)
   //	hipEventRecord(streamSync[i], cudaStreams[i]);
   //hipEventRecord(pmeStreamSync, pmeStream);

   //for (int i = 0; i < 5; i++) {
   //	if (i != 0)
   //		hipStreamWaitEvent(cudaStreams[0], streamSync[i], 0);
   //	if (i != 1)
   //		hipStreamWaitEvent(cudaStreams[1], streamSync[i], 0);
   //}
   //hipStreamWaitEvent(cudaStreams[0], pmeStreamSync, 0);// solvents dont need to wait for pme, as they are only neutral currently
   //hipStreamWaitEvent(cudaStreams[1], pmeStreamSync, 0);// solvents dont need to wait for pme, as they are only neutral currently

   //// #### Integration and Transfer kernels

   //if (boxparams.n_compounds > 0) {
   //	CompoundIntegrationKernel<BoundaryCondition, emvariant> 
   //		<<<boxparams.n_compounds, MAX_COMPOUND_PARTICLES, 0, cudaStreams[0] >> >
   //		(sim_dev, compoundForceEnergyInterims, forceEnergiesBondgroups, forceEnergiesPME);
   //}

   //if (boxparams.n_solvents > 0) {		
   //	TinymolIntegrationLoggingAndTransferout<BoundaryCondition, emvariant>
   //		<< <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, 0, cudaStreams[1] >> >
   //		(sim_dev);
   //	solventTransferKernel<BoundaryCondition>
   //		<< <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlockTransfermodule::max_queue_size, 0, cudaStreams[1] >> >
   //		(sim_dev);
   //}	


   //// End graph capture
   //auto c = hipStreamEndCapture(masterStream, &graph);

   //for (int i = 0; i < 5; i++)
   //	hipEventDestroy(streamSync[i]);
   //hipEventDestroy(pmeStreamSync);

   //for (hipStream_t& stream : cudaStreams) {
   //	hipStreamDestroy(stream);
   //}
   //hipStreamDestroy(pmeStream);