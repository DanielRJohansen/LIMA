#include "hip/hip_runtime.h"
#include "Engine.cuh"
#include "Utilities.h"
#include "Neighborlists.cuh"
#include "Statistics.h"

#include "BoundaryCondition.cuh"
#include "EngineBodies.cuh"
#include "SimulationDevice.cuh"
#include "LimaPositionSystem.cuh"

#include "ChargeOcttree.cuh"

#include "EngineKernels.cuh"

#include "Thermostat.cuh"

#include "SupernaturalForces.cuh"


#include <unordered_set>

//const int compound_size = sizeof(CompoundCompact);
//const int nlsit_size = sizeof(NeighborList);
//const int sssize = (sizeof(Float3) + sizeof(float)) * THREADS_PER_COMPOUNDBLOCK;
//const int Ckernel_shared_mem = sizeof(CompoundCompact) + sizeof(NeighborList) +
//	(2* sizeof(Float3)) * THREADS_PER_COMPOUNDBLOCK + sizeof(Coord) + sizeof(Float3) + clj_utilitybuffer_bytes;
//static_assert(Ckernel_shared_mem < 45000, "Not enough shared memory for CompoundKernel");

//const int sbsize = sizeof(SolventBlock);
//const int Skernel_shared_mem = (sizeof(Float3) + 1) * SolventBlock::MAX_SOLVENTS_IN_BLOCK + sizeof(SolventBlock)
//	+ sizeof(SolventTransferqueue<SolventBlockTransfermodule::max_queue_size>) * 6
//	+ 4 + 4 * 3 * 2;

Engine::Engine(std::unique_ptr<Simulation> sim, BoundaryConditionSelect bc, std::unique_ptr<LimaLogger> logger)
	: bc_select(bc), m_logger(std::move(logger))
{
	simulation = std::move(sim);

	verifyEngine();

	dataBuffersDevice = std::make_unique<DatabuffersDeviceController>(simulation->box_host->boxparams.total_particles_upperbound, 
		simulation->box_host->boxparams.n_compounds, simulation->simparams_host.data_logging_interval);


	// Create the Sim_dev {
	{
		if (sim_dev != nullptr) { throw std::runtime_error("Expected simdev to be null to move sim to device"); }
		sim_dev = new SimulationDevice(simulation->simparams_host, simulation->box_host.get(), BoxConfig::Create(*simulation->box_host), BoxState::Create(*simulation->box_host), *dataBuffersDevice);
		sim_dev = genericMoveToDevice(sim_dev, 1);
	}
	setDeviceConstantMemory();
	boxStateCopy = std::make_unique<BoxState>(nullptr, nullptr, nullptr, nullptr, nullptr);
	boxConfigCopy = std::make_unique<BoxConfig>(nullptr, nullptr, nullptr, nullptr, nullptr);
	hipMemcpy(boxStateCopy.get(), sim_dev->boxState, sizeof(BoxState), hipMemcpyDeviceToHost);
	hipMemcpy(boxConfigCopy.get(), &sim_dev->boxConfig, sizeof(BoxConfig), hipMemcpyDeviceToHost);
	neighborlistsPtr = sim_dev->compound_neighborlists;





	//std::unordered_set<std::string> unique_compounds;
	//for (int i = 0; i < simulation->box_host->boxparams.n_compounds; i++) {
	//	char types[64];
	//	memcpy(types, simulation->box_host->compounds[i].atom_types, 64);
	//	unique_compounds.insert(std::string(types));
	//}
	//int a = 0;






	auto boxparams = simulation->box_host->boxparams;
	thermostat = std::make_unique<Thermostat>(boxparams.n_compounds, boxparams.n_solvents, boxparams.total_particles_upperbound);

	// To create the NLists we need to bootstrap the traj_buffer, since it has no data yet
	bootstrapTrajbufferWithCoords();

	NeighborLists::updateNlists(sim_dev, simulation->getStep(), simulation->simparams_host.bc_select, simulation->box_host->boxparams, timings.nlist);
	m_logger->finishSection("Engine Ready");
}

Engine::~Engine() {
	if (sim_dev != nullptr) {
		sim_dev->FreeMembers();
		hipFree(sim_dev);
	}

	assert(simulation == nullptr);
}


void Engine::setDeviceConstantMemory() {
	//const int forcefield_bytes = sizeof(ForceField_NB);
	hipMemcpyToSymbol(HIP_SYMBOL(forcefield_device), &simulation->forcefield, sizeof(ForceField_NB), 0, hipMemcpyHostToDevice);	// So there should not be a & before the device __constant__
	hipMemcpyToSymbol(HIP_SYMBOL(tinymolForcefield_device), &simulation->forcefieldTinymol, sizeof(ForcefieldTinymol), 0, hipMemcpyHostToDevice);

	BoxSize boxSize_host;
	boxSize_host.Set(simulation->box_host->boxparams.boxSize);
	hipMemcpyToSymbol(HIP_SYMBOL(boxSize_device), &boxSize_host, sizeof(BoxSize), 0, hipMemcpyHostToDevice);
	//SetConstantMem(simulation->boxparams_host.boxSize);
	//BoxSize bs;
	//hipMemcpyFromSymbol(&bs, HIP_SYMBOL(boxSize_device), sizeof(BoxSize));

//	hipDeviceSynchronize();


	//BoxSize bs1;
	//hipMemcpyFromSymbol(&bs1, HIP_SYMBOL(boxSize_device), sizeof(BoxSize));

	hipMemcpyToSymbol(HIP_SYMBOL(cutoffNm_device), &simulation->simparams_host.cutoff_nm, sizeof(float), 0, hipMemcpyHostToDevice);
	const float cutoffLmSquaredReciprocal = 1.f / (simulation->simparams_host.cutoff_nm * NANO_TO_LIMA * simulation->simparams_host.cutoff_nm * NANO_TO_LIMA);
	hipMemcpyToSymbol(HIP_SYMBOL(cutoffLmSquaredReciprocal_device), &cutoffLmSquaredReciprocal, sizeof(float), 0, hipMemcpyHostToDevice);


	const float initialThermostatScalar = 1.f;
	hipMemcpyToSymbol(HIP_SYMBOL(thermostatScalar_device), &initialThermostatScalar, sizeof(float), 0, hipMemcpyHostToDevice);

	LIMA_UTILS::genericErrorCheck("Error while setting Global Constants\n");
}


std::unique_ptr<Simulation> Engine::takeBackSim() {
	assert(sim_dev);
	sim_dev->boxState->CopyDataToHost(*simulation->box_host);
	return std::move(simulation);
}

void Engine::verifyEngine() {
	LIMA_UTILS::genericErrorCheck("Error before engine initialization.\n");

	const int nBlocks = simulation->box_host->boxparams.boxSize;
	assert(nBlocks* nBlocks* nBlocks < INT32_MAX && "Neighborlist cannot handle such large gridnode_ids");

	if constexpr (ENABLE_ES_LR) {
		if (simulation->simparams_host.enable_electrostatics && simulation->simparams_host.bc_select != PBC) {
			throw std::invalid_argument("Electrostatics only supported with PBC at the current time");
		}
	}
}

void Engine::step() {
	LIMA_UTILS::genericErrorCheck("Error before step!");

	deviceMaster();	// Device first, otherwise offloading data always needs the last datapoint!
	assert(simulation);
	assert(sim_dev);
	simulation->step++;

	hostMaster();

	LIMA_UTILS::genericErrorCheck("Error after step!");
}

void Engine::hostMaster() {						// This is and MUST ALWAYS be called after the deviceMaster, and AFTER incStep()!
	auto t0 = std::chrono::high_resolution_clock::now();
	if (DatabuffersDeviceController::IsBufferFull(simulation->getStep(), simulation->simparams_host.data_logging_interval)) {
		offloadLoggingData(DatabuffersDeviceController::nStepsInBuffer);
		runstatus.stepForMostRecentData = simulation->getStep();

		if ((simulation->getStep() % simulation->simparams_host.steps_per_temperature_measurement) == 0 && simulation->getStep() > 0) {
			auto [temperature, thermostatScalar] = thermostat->Temperature(sim_dev, simulation->box_host->boxparams, simulation->simparams_host);
			simulation->temperature_buffer.push_back(temperature);
			runstatus.current_temperature = temperature;

			if (simulation->simparams_host.apply_thermostat)
				hipMemcpyToSymbol(HIP_SYMBOL(thermostatScalar_device), &thermostatScalar, sizeof(float), 0, hipMemcpyHostToDevice);
		}
		
		HandleEarlyStoppingInEM();

		NeighborLists::updateNlists(sim_dev, simulation->getStep(), simulation->simparams_host.bc_select, simulation->box_host->boxparams, timings.nlist);
	}

	// Handle status
	runstatus.current_step = simulation->getStep();
	runstatus.critical_error_occured = sim_dev->signals->critical_error_encountered;	// TODO: Can i get this from simparams_host? UNSAFE
	if (runstatus.current_step >= simulation->simparams_host.n_steps || runstatus.critical_error_occured)
		runstatus.simulation_finished = true;


	const auto t1 = std::chrono::high_resolution_clock::now();
	const int cpu_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
	timings.cpu_master += cpu_duration;
}

void Engine::terminateSimulation() {
	const int64_t stepsReadyToTransfer = DatabuffersDeviceController::StepsReadyToTransfer(simulation->getStep(), simulation->simparams_host.data_logging_interval);
	offloadLoggingData(stepsReadyToTransfer);

	sim_dev->boxState->CopyDataToHost(*simulation->box_host);

	LIMA_UTILS::genericErrorCheck("Error during TerminateSimulation");
}

//--------------------------------------------------------------------------	CPU workload --------------------------------------------------------------//

void Engine::offloadLoggingData(const int64_t steps_to_transfer) {
	assert(steps_to_transfer <= simulation->getStep());
	if (steps_to_transfer == 0) { return; }


	const int64_t startstep = simulation->getStep() - steps_to_transfer * simulation->simparams_host.data_logging_interval;
	const int64_t startindex = LIMALOGSYSTEM::getMostRecentDataentryIndex(startstep, simulation->simparams_host.data_logging_interval);
	const int64_t indices_to_transfer = LIMALOGSYSTEM::getNIndicesBetweenSteps(startstep, simulation->getStep(), simulation->simparams_host.data_logging_interval);
	const int particlesUpperbound = simulation->box_host->boxparams.total_particles_upperbound;
	hipMemcpyAsync(
		simulation->potE_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->potE_buffer,
		sizeof(float) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->vel_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->vel_buffer,
		sizeof(float) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->forceBuffer->getBufferAtIndex(startindex),
		dataBuffersDevice->forceBuffer,
		sizeof(Float3) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	hipMemcpyAsync(
		simulation->traj_buffer->getBufferAtIndex(startindex),
		dataBuffersDevice->traj_buffer,
		sizeof(Float3) * particlesUpperbound * indices_to_transfer,
		hipMemcpyDeviceToHost);

	step_at_last_traj_transfer = simulation->getStep();
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(LIMALOGSYSTEM::getMostRecentDataentryIndex(simulation->getStep() - 1, simulation->simparams_host.data_logging_interval));

	hipDeviceSynchronize();
}

void Engine::offloadTrainData() {
#ifdef GENERATETRAINDATA
	uint64_t values_per_step = N_DATAGAN_VALUES * MAX_COMPOUND_PARTICLES * simulation->boxparams_host.n_compounds;
	if (values_per_step == 0) {
		return;	// No data to transfer
	}

	uint64_t step_offset = (simulation->getStep() - STEPS_PER_TRAINDATATRANSFER) * values_per_step;	// fix max_compound to the actual count save LOTS of space!. Might need a file in simout that specifies cnt for loading in other programs...
	hipMemcpy(&simulation->trainingdata[step_offset], dataBuffersDevice->data_GAN, sizeof(Float3) * values_per_step * STEPS_PER_TRAINDATATRANSFER, hipMemcpyDeviceToHost);
	LIMA_UTILS::genericErrorCheck("Cuda error during traindata offloading\n");
#endif
}


void Engine::bootstrapTrajbufferWithCoords() {
	if (simulation->simparams_host.n_steps == 0) return;

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");

	// We need to bootstrap step-0 which is used for traj-buffer
	for (int compound_id = 0; compound_id < simulation->box_host->boxparams.n_compounds; compound_id++) {
		for (int particle_id = 0; particle_id < MAX_COMPOUND_PARTICLES; particle_id++) {
			const Float3 particle_abspos = LIMAPOSITIONSYSTEM::GetAbsolutePositionNM(simulation->box_host->compoundCoordsBuffer[compound_id].origo, simulation->box_host->compoundCoordsBuffer[compound_id].rel_positions[particle_id]);
			simulation->traj_buffer->getCompoundparticleDatapointAtIndex(compound_id, particle_id, 0) = particle_abspos;
		}
	}

	step_at_last_traj_transfer = 0.f;
	runstatus.most_recent_positions = simulation->traj_buffer->getBufferAtIndex(0);

	LIMA_UTILS::genericErrorCheck("Error during bootstrapTrajbufferWithCoords");
}

void Engine::HandleEarlyStoppingInEM() {
	if (!simulation->simparams_host.em_variant || simulation->getStep() == simulation->simparams_host.n_steps)
		return;
	
	const int minStepsPerCheck = 100;
	if (simulation->getStep() > stepAtLastEarlystopCheck + minStepsPerCheck) {
		const float greatestForce = Statistics::MaxLen(simulation->forceBuffer->GetBufferAtStep(simulation->getStep()-1), simulation->forceBuffer->EntriesPerStep());
		runstatus.greatestForce = greatestForce / LIMA * NANO / KILO; // Convert to [kJ/mol/nm]
		simulation->maxForceBuffer.emplace_back(std::pair<int64_t,float>{ simulation->getStep(), runstatus.greatestForce });

		if (runstatus.greatestForce <= simulation->simparams_host.em_force_tolerance) {
			runstatus.simulation_finished = true;
		}

		stepAtLastEarlystopCheck = simulation->getStep();
	}
}




//--------------------------------------------------------------------------	SIMULATION BEGINS HERE --------------------------------------------------------------//
template <typename BoundaryCondition, bool emvariant, bool computePotE>
void Engine::_deviceMaster() {
	const auto t0 = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();

	const BoxParams& boxparams = simulation->box_host->boxparams;

	const bool logData = simulation->getStep() % simulation->simparams_host.data_logging_interval == 0;// TODO maybe log at the final step, not 0th?

	if (boxparams.n_compounds > 0) {
		compoundFarneighborShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE> << <boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK >> > (simulation->getStep(), *boxStateCopy, *boxConfigCopy, neighborlistsPtr, simulation->simparams_host.enable_electrostatics);
		//LAUNCH_GENERIC_KERNEL_3(compoundFarneighborShortrangeInteractionsKernel, boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, simulation->simparams_host.em_variant, logData, sim_dev, simulation->getStep());
	}

	LIMA_UTILS::genericErrorCheck("Error after compoundForceKernel");

	if (boxparams.n_compounds > 0) {
		//LAUNCH_GENERIC_KERNEL_3(compoundLJKernel, boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, simulation->simparams_host.em_variant, logData, sim_dev, simulation->getStep());		
		//LAUNCH_GENERIC_KERNEL_3(compoundImmediateneighborAndSelfShortrangeInteractionsKernel, boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, simulation->simparams_host.em_variant, logData, sim_dev, simulation->getStep());
		compoundImmediateneighborAndSelfShortrangeInteractionsKernel<BoundaryCondition, emvariant, computePotE> << <boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK >> > (sim_dev, simulation->getStep());
	}

	LIMA_UTILS::genericErrorCheck("Error after compoundForceKernel");


	const auto t0a = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();

	if constexpr (ENABLE_ES_LR) {
		if (simulation->simparams_host.enable_electrostatics) {
			timings.electrostatics += Electrostatics::HandleElectrostatics(sim_dev, boxparams);
		}
	}
	const auto t0b = std::chrono::high_resolution_clock::now();

	if (boxparams.n_bridges > 0) {
		//LAUNCH_GENERIC_KERNEL(compoundBridgeKernel, boxparams.n_bridges, MAX_PARTICLES_IN_BRIDGE, bc_select, sim_dev, simulation->getStep());
		compoundBridgeKernel<BoundaryCondition> << <boxparams.n_bridges, MAX_PARTICLES_IN_BRIDGE >> > (sim_dev, simulation->getStep());
	}

	if (simulation->simparams_host.snf_select != None) {
		SupernaturalForces::SnfHandler(simulation.get(), sim_dev, simulation->getStep());
	}

	hipDeviceSynchronize();
	if (boxparams.n_compounds > 0) {
		//LAUNCH_GENERIC_KERNEL_2(compoundBondsAndIntegrationKernel, boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, simulation->simparams_host.em_variant, sim_dev, simulation->getStep());
		compoundBondsAndIntegrationKernel<BoundaryCondition, emvariant> << <boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK >> > (sim_dev, simulation->getStep(), simulation->box_host->uniformElectricField);
	}
	LIMA_UTILS::genericErrorCheck("Error after compoundForceKernel");
	const auto t1 = std::chrono::high_resolution_clock::now();


#ifdef ENABLE_SOLVENTS
	if (boxparams.n_solvents > 0) {
		//LAUNCH_GENERIC_KERNEL_2(solventForceKernel, BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK, bc_select, simulation->simparams_host.em_variant, sim_dev, simulation->getStep());
		solventForceKernel<BoundaryCondition, emvariant> << <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlock::MAX_SOLVENTS_IN_BLOCK >> > (sim_dev, simulation->getStep());


		hipDeviceSynchronize();
		LIMA_UTILS::genericErrorCheck("Error after solventForceKernel");
		if (SolventBlocksCircularQueue::isTransferStep(simulation->getStep())) {
			//LAUNCH_GENERIC_KERNEL(solventTransferKernel, BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlockTransfermodule::max_queue_size, bc_select, sim_dev, simulation->getStep());
			solventTransferKernel<BoundaryCondition> << <BoxGrid::BlocksTotal(BoxGrid::NodesPerDim(boxparams.boxSize)), SolventBlockTransfermodule::max_queue_size >> > (sim_dev, simulation->getStep());
		}
	}
	hipDeviceSynchronize();
	LIMA_UTILS::genericErrorCheck("Error after solventTransferKernel");
#endif

	if (boxparams.n_compounds > 0) {
		//LAUNCH_GENERIC_KERNEL_2(CompoundIntegrationKernel, boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK, bc_select, simulation->simparams_host.em_variant, sim_dev, simulation->getStep());
		CompoundIntegrationKernel<BoundaryCondition, emvariant> << <boxparams.n_compounds, THREADS_PER_COMPOUNDBLOCK >> > (sim_dev, simulation->getStep());
	}
	LIMA_UTILS::genericErrorCheck("Error after CompoundIntegrationKernel");

	const auto t2 = std::chrono::high_resolution_clock::now();

	const int compounds_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0b + t0a - t0).count();
	const int solvents_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();

	timings.compound_kernels += compounds_duration;
	timings.solvent_kernels += solvents_duration;
}
template void Engine::_deviceMaster<PeriodicBoundaryCondition, true, true>();
template void Engine::_deviceMaster<PeriodicBoundaryCondition, true, false>();
template void Engine::_deviceMaster<PeriodicBoundaryCondition, false, true>();
template void Engine::_deviceMaster<PeriodicBoundaryCondition, false, false>();
template void Engine::_deviceMaster<NoBoundaryCondition, true, true>();
template void Engine::_deviceMaster<NoBoundaryCondition, true, false>();
template void Engine::_deviceMaster<NoBoundaryCondition, false, true>();
template void Engine::_deviceMaster<NoBoundaryCondition, false, false>();



void Engine::deviceMaster() {

	const bool logData = simulation->getStep() % simulation->simparams_host.data_logging_interval == 0;// TODO maybe log at the final step, not 0th?

	switch (simulation->simparams_host.bc_select) {
	case NoBC:
		if (simulation->simparams_host.em_variant) {
			if (logData) {
				_deviceMaster<NoBoundaryCondition, true, true>();
			}
			else {
				_deviceMaster<NoBoundaryCondition, true, false>();
			}
		}
		else {
			if (logData) {
				_deviceMaster<NoBoundaryCondition, false, true>();
			}
			else {
				_deviceMaster<NoBoundaryCondition, false, false>();
			}
		}
		break;
	case PBC:
		if (simulation->simparams_host.em_variant) {
			if (logData) {
				_deviceMaster<PeriodicBoundaryCondition, true, true>();
			}
			else {
				_deviceMaster<PeriodicBoundaryCondition, true, false>();
			}
		}
		else {
			if (logData) {
				_deviceMaster<PeriodicBoundaryCondition, false, true>();
			}
			else {
				_deviceMaster<PeriodicBoundaryCondition, false, false>();
			}
		}
		break;
	default:
		throw std::runtime_error("Unsupported boundary condition in LAUNCH_GENERIC_KERNEL");
	}
}
