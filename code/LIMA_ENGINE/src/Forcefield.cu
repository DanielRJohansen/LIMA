#include "hip/hip_runtime.h"
#pragma once

#include "Forcefield.cuh"
#include "Printer.h"

using namespace LIMA_Print;


Forcefield::Forcefield(VerbosityLevel vl) : vl(vl) {};

void Forcefield::loadForcefield(string molecule_dir) {
	if (vl >= CRITICAL_INFO) { printH2("Building forcefield"); }

	vector<vector<string>> summary_rows = Filehandler::readFile(molecule_dir + "/LIMA_ffnonbonded_filtered.txt", INT_MAX, vl >= V2);
	vector<vector<string>> forcefield_rows = Filehandler::readFile(molecule_dir + "/LIMA_ffbonded_filtered.txt", INT_MAX, vl >= V2);


	nb_atomtypes = parseAtomTypes(summary_rows);					// 1 entry per type in compressed forcefield
	loadAtomypesIntoForcefield();


	nb_atomtype_ids = parseAtomTypeIDs(forcefield_rows);				// 1 entry per atom in conf

	topol_bonds = parseBonds(forcefield_rows);
	topol_angles = parseAngles(forcefield_rows);
	topol_dihedrals = parseDihedrals(forcefield_rows);

	if (vl >= CRITICAL_INFO) {
		printf("Nonbonded parameters size: %llu bytes\n", sizeof(ForceField_NB));
		printH2("Finished building forcefield");
	}
}


int Forcefield::getAtomtypeID(int global_id) {
	if (global_id > n_atoms || global_id == 0) {	// 0 is an error, as atoms are 1-indexed
		printf("Attempting to fetch atomtype of non-loaded atom with global_id %d\n", global_id);
		exit(0);
	}
	return nb_atomtype_ids[global_id];
}

PairBond* Forcefield::getBondType(int id1, int id2) {
	for (int i = 0; i < n_topol_bonds; i++) {
		if (topol_bonds[i].atom_indexes[0] == id1 && topol_bonds[i].atom_indexes[1] == id2) {
			return &topol_bonds[i];
		}
	}
	printf("Bond not found with ids %d %d\n", id1, id2);
	exit(0);
}

AngleBond* Forcefield::getAngleType(int id1, int id2, int id3) {
	for (int i = 0; i < n_topol_angles; i++) {
		if (topol_angles[i].atom_indexes[0] == id1 && topol_angles[i].atom_indexes[1] == id2 && topol_angles[i].atom_indexes[2] == id3) {
			return &topol_angles[i];
		}
	}
	printf("Angle not found with ids %d %d %d\n", id1, id2, id3);
	exit(0);
}

DihedralBond* Forcefield::getDihedralType(int id1, int id2, int id3, int id4) {
	for (int i = 0; i < n_topol_dihedrals; i++) {
		if (topol_dihedrals[i].atom_indexes[0] == id1 && topol_dihedrals[i].atom_indexes[1] == id2 && topol_dihedrals[i].atom_indexes[2] == id3 && topol_dihedrals[i].atom_indexes[3] == id4) {
			return &topol_dihedrals[i];
		}
	}
	printf("Dihedral not found with ids %d %d %d %d\n", id1, id2, id3, id4);
	exit(0);
}












NBAtomtype* Forcefield::parseAtomTypes(vector<vector<string>> summary_rows) {
	NBAtomtype* atomtypes = new NBAtomtype[10000];
	int ptr = 0;
	STATE current_state = INACTIVE;

	for (vector<string> row : summary_rows) {
		if (newParseTitle(row)) {
			current_state = setState(row[1], current_state);
			continue;
		}

		

		if (current_state == FF_NONBONDED) {
			//for (string e : row)
				//cout << e << '\t';
			//printf("\n");
			// Row is type, id, weight [g], sigma [nm], epsilon [J/mol]
			atomtypes[ptr++] = NBAtomtype(stof(row[2]), stof(row[3]), stof(row[4]));
		}			
	}
	n_nb_atomtypes = ptr;
	if (vl >= V1) { printf("%d NB_Atomtypes loaded\n", ptr); }
	return atomtypes;
}

int* Forcefield::parseAtomTypeIDs(vector<vector<string>> forcefield_rows) {	// returns the nonbonded atomtype
	int* atomtype_ids = new int[10000];
	STATE current_state = INACTIVE;

	for (vector<string> row : forcefield_rows) {
		if (newParseTitle(row)) {
			current_state = setState(row[1], current_state);
			continue;
		}

		if (current_state == NB_ATOMTYPES) {
			atomtype_ids[stoi(row[0])] = stoi(row[1]);
			n_atoms++;
		}
			
	}
	if (vl >= V1) { printf("%d NB_Atomtype_IDs loaded\n", n_atoms); }
	return atomtype_ids;
}

PairBond* Forcefield::parseBonds(vector<vector<string>> forcefield_rows) {
	PairBond* bonds = new PairBond[10000];
	int ptr = 0;
	STATE current_state = INACTIVE;

	for (vector<string> row : forcefield_rows) {
		if (newParseTitle(row)) {
			current_state = setState(row[1], current_state);
			continue;
		}

		if (current_state == BONDS) {
			bonds[ptr++] = PairBond(stoi(row[0]), stoi(row[1]), stof(row[4]), stof(row[5]));
		}
	}
	n_topol_bonds = ptr;
	if (vl >= V1) { printf("%d bonds loaded\n", ptr); }
	return bonds;
}

AngleBond* Forcefield::parseAngles(vector<vector<string>> forcefield_rows) {
	AngleBond* angles = new AngleBond[10000];
	int ptr = 0;
	STATE current_state = INACTIVE;

	for (vector<string> row : forcefield_rows) {
		if (newParseTitle(row)) {
			current_state = setState(row[1], current_state);
			continue;
		}

		if (current_state == ANGLES) {
			angles[ptr++] = AngleBond(stoi(row[0]), stoi(row[1]), stoi(row[2]), stof(row[6]) , stof(row[7]));		// Assumes radians here
		}

	}
	n_topol_angles = ptr;
	if (vl >= V1) { printf("%d angles loaded\n", ptr); }
	return angles;
}

DihedralBond* Forcefield::parseDihedrals(vector<vector<string>> forcefield_rows) {
	DihedralBond* dihedrals = new DihedralBond[10000];
	int ptr = 0;
	STATE current_state = INACTIVE;

	for (vector<string> row : forcefield_rows) {
		if (newParseTitle(row)) {
			current_state = setState(row[1], current_state);
			
		//	if (has_been_enabled)	// To deal with the wierd dihedrals at the bottom of the topol.top
			//	break;
			continue;
		}

		if (current_state == DIHEDRALS) {
			dihedrals[ptr++] = DihedralBond(stoi(row[0]), stoi(row[1]), stoi(row[2]), stoi(row[3]), stof(row[8]), abs(stof(row[9])), stoi(row[10]));			// MIGHT HAVE TO DO AN ABS() ON K_PHI, SINCE IT IS NEGATIVE SOMETIMES??? WHAT THE FUCKKKKKKKKKK CHEMISTS?????!?!?!
			//has_been_enabled = true;
		}
	}
	n_topol_dihedrals = ptr;
	if (vl >= V1) { printf("%d dihedrals loaded\n", ptr); }
	return dihedrals;
}






void Forcefield::loadAtomypesIntoForcefield() {
	static const float mass_min = 0.001f;	// [kg/mol]
	static const float sigma_min = 0.001f;
	static const float epsilon_min = 0.001f;

	for (int i = 0; i < n_nb_atomtypes; i++) {
		forcefield.particle_parameters[i].mass = nb_atomtypes[i].mass * 1e-3f;		// Convert g/mol to kg/mol
		forcefield.particle_parameters[i].sigma = nb_atomtypes[i].sigma / NORMALIZER;	// Convert to normalized value
		forcefield.particle_parameters[i].epsilon = nb_atomtypes[i].epsilon / (NORMALIZER * NORMALIZER);

		bool illegal_parameter = (forcefield.particle_parameters[i].mass < mass_min) || (forcefield.particle_parameters[i].sigma < sigma_min) || (forcefield.particle_parameters[i].epsilon < epsilon_min);

		if ((vl >= V2) || illegal_parameter) { printf("Mass %f Sigma %f Epsilon %f\n", nb_atomtypes[i].mass, nb_atomtypes[i].sigma, nb_atomtypes[i].epsilon); }
	}
}
