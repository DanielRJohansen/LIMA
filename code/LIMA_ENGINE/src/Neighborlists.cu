#include "Neighborlists.cuh"
#include <algorithm>
#include <execution>


bool neighborWithinCutoff(const Float3* pos_a, const Float3* pos_b, const float cutoff_lm) {		// This is used for compounds with a confining_particle_sphere from key_particle BEFORE CUTOFF begins
	const float dist = EngineUtils::calcHyperDist(pos_a, pos_b);
	return dist < cutoff_lm;
}

void inline addNeighborIfEligible(HashTable& currentNeighbors,
	NeighborList& nlist_self, NeighborList& nlist_other,
	const Float3& pos_self, const Float3& pos_other,
	const int& id_self, const int& id_other,
	NeighborList::NEIGHBOR_TYPE neighbortype_self, NeighborList::NEIGHBOR_TYPE neighbortype_other,
	const float cutoff_extension)
{
	if (neighborWithinCutoff(&pos_self, &pos_other, CUTOFF_LM + cutoff_extension)) {
		if (currentNeighbors.insert(id_other)) {
			nlist_self.addId(id_other, neighbortype_other);
			nlist_other.addId(id_self, neighbortype_self);
		}
	}
}



inline NListDataCollection::NListDataCollection(Simulation* simulation) {
	n_compounds = simulation->n_compounds;
	n_solvents = simulation->n_solvents;
	compoundstates = new CompoundState[n_compounds];
	//solvents = new Solvent[simulation->n_solvents];
	compound_neighborlists = new NeighborList[MAX_COMPOUNDS];
	solvent_neighborlists = new NeighborList[MAX_SOLVENTS];
	hipMemcpy(compound_neighborlists, simulation->box->compound_neighborlists, sizeof(NeighborList) * n_compounds, hipMemcpyDeviceToHost);
	hipMemcpy(solvent_neighborlists, simulation->box->solvent_neighborlists, sizeof(NeighborList) * n_solvents, hipMemcpyDeviceToHost);
}

void NListDataCollection::preparePositionData(const Simulation& simulation, const uint32_t step_at_update) {
	auto step = step_at_update;

	// Data for the current step has not yet been generated so we need to use the previous step.
	// For the very first step, engine has cheated and already written the traj from the initial setup.	
	if (step != 0) { step--; }

	for (int compound_id = 0; compound_id < n_compounds; compound_id++) {
		const size_t index = EngineUtils::getAlltimeIndexOfParticle(step, simulation.total_particles_upperbound, compound_id, 0);
		compound_key_positions[compound_id] = simulation.traj_buffer[index];
	}

	// TODO: we should probably apply PBC here, to avoid problems with the blocks...
	for (int solvent_id = 0; solvent_id < n_solvents; solvent_id++) {
		const size_t index = EngineUtils::getAlltimeIndexOfParticle(step, simulation.total_particles_upperbound, simulation.n_compounds, solvent_id);
		solvent_positions[solvent_id] = simulation.traj_buffer[index];
	}
}




NListManager::NListManager(Simulation* simulation) {
	nlist_data_collection = new NListDataCollection(simulation);


	for (int i = 0; i < nlist_data_collection->n_compounds; i++) {
		nlist_data_collection->compound_neighborlists[i].associated_id = i;
	}
	for (int i = 0; i < nlist_data_collection->n_solvents; i++) {
		nlist_data_collection->solvent_neighborlists[i].associated_id = i;
	}
}

// Main sim thread enters this block, so make sure it can leave VERY quickly
void NListManager::updateNeighborLists(Simulation* simulation, bool* updatenlists_mutexlock, bool force_update, bool async, int* timings, bool* critical_error) {
	const uint32_t step_at_update = simulation->getStep();

	if (async && !force_update) {
		std::thread nlist_worker(NListUtils::updateNeighborLists, simulation, nlist_data_collection, &updated_neighborlists_ready, timings, updatenlists_mutexlock, step_at_update);
		nlist_worker.detach();
	}
	else {
		NListUtils::updateNeighborLists(simulation, nlist_data_collection, &updated_neighborlists_ready, timings, updatenlists_mutexlock, step_at_update);
	}

	prev_update_step = step_at_update;

	if (force_update) {
		Int3 n_data(nlist_data_collection->compound_neighborlists[0].n_compound_neighbors, nlist_data_collection->compound_neighborlists[0].n_solvent_neighbors, 0);
		printf("\nEntity neighbors: %d %d\n", n_data.x, n_data.y);
	}
}


void NListManager::pushNlistsToDevice(Simulation* simulation) {
	hipMemcpy(simulation->box->compound_neighborlists, nlist_data_collection->compound_neighborlists, sizeof(NeighborList) * simulation->n_compounds, hipMemcpyHostToDevice);
	hipMemcpy(simulation->box->solvent_neighborlists, nlist_data_collection->solvent_neighborlists, sizeof(NeighborList) * simulation->n_solvents, hipMemcpyHostToDevice);
	updated_neighborlists_ready = 0;
}








namespace NListUtils {
	void cullDistantNeighbors(Simulation* simulation, NListDataCollection* nlist_data_collection) {
		for (int id_self = 0; id_self < nlist_data_collection->n_compounds; id_self++) {
			NeighborList* nlist_self = &nlist_data_collection->compound_neighborlists[id_self];
			float cutoff_add_self = simulation->compounds_host[id_self].confining_particle_sphere;



			for (int j = 0; j < nlist_self->n_compound_neighbors; j++) {		// Cull compound-compound
				int id_neighbor = nlist_self->neighborcompound_ids[j];
				NeighborList* nlist_neighbor = &nlist_data_collection->compound_neighborlists[id_neighbor];
				float cutoff_add_neighbor = simulation->compounds_host[id_neighbor].confining_particle_sphere;

				if (id_self < id_neighbor) {
					if (!neighborWithinCutoff(&nlist_data_collection->compound_key_positions[id_self], &nlist_data_collection->compound_key_positions[id_neighbor], cutoff_add_self + cutoff_add_neighbor + CUTOFF_LM)) {
						nlist_self->removeId(id_neighbor, NeighborList::NEIGHBOR_TYPE::COMPOUND);
						nlist_neighbor->removeId(id_self, NeighborList::NEIGHBOR_TYPE::COMPOUND);
						j--;	// Decrement, as the removeId puts the last element at the current and now vacant spot.
					}
				}
			}


			for (int j = 0; j < nlist_self->n_solvent_neighbors; j++) {			// Cull compound-solvent
				int id_neighbor = nlist_self->neighborsolvent_ids[j];
				NeighborList* nlist_neighbor = &nlist_data_collection->solvent_neighborlists[id_neighbor];

				//printf("Dist: %f\n", (nlist_data_collection->compound_key_positions[id_self] - nlist_data_collection->solvent_positions[id_neighbor]).len());
				if (!neighborWithinCutoff(&nlist_data_collection->compound_key_positions[id_self], &nlist_data_collection->solvent_positions[id_neighbor], cutoff_add_self + CUTOFF_LM) && false) {
					nlist_self->removeId(id_neighbor, NeighborList::NEIGHBOR_TYPE::SOLVENT);
					//	printf("J: %d\n", j);
					nlist_neighbor->removeId(id_self, NeighborList::NEIGHBOR_TYPE::COMPOUND);
					j--;	// Decrement, as the removeId puts the last element at the current and now va-cant spot.
				}
			}
		}


		for (int id_self = 0; id_self < nlist_data_collection->n_solvents; id_self++) {																// Cull solvent-solvent
			NeighborList* nlist_self = &nlist_data_collection->solvent_neighborlists[id_self];

			int cnt = 0;

			for (int j = 0; j < nlist_self->n_solvent_neighbors; j++) {			/// NOT FINISHED HERE
				int id_neighbor = nlist_self->neighborsolvent_ids[j];
				NeighborList* nlist_neighbor = &nlist_data_collection->solvent_neighborlists[id_neighbor];

				if (!neighborWithinCutoff(&nlist_data_collection->solvent_positions[id_self], &nlist_data_collection->solvent_positions[id_neighbor], CUTOFF_LM)) {
					cnt++;
					if (!nlist_self->removeId(id_neighbor, NeighborList::NEIGHBOR_TYPE::SOLVENT))
						printf("J1: %d id_self %d id_neighbor %d    cnt %d\n", j, id_self, id_neighbor, cnt);
					if (!nlist_neighbor->removeId(id_self, NeighborList::NEIGHBOR_TYPE::SOLVENT)) {
						printf("J2: %d of %d.   id_self %d id_neighbor %d count: %d\n", j, nlist_self->n_solvent_neighbors, id_self, id_neighbor, cnt);
						for (int i = 0; i < nlist_self->n_solvent_neighbors; i++) {
							printf("neighbor %d\n", nlist_self->neighborsolvent_ids[i]);
						}
						printf("\n\n\n");
						exit(1);
					}


					j--;	// Decrement, as the removeId puts the last element at the current and now vacant spot.
				}
			}
		}
	}

	// Important: do NOT call getStep during this funciton, as it runs async!!!!
	// This is a thread worker-function, so it can't own the object, thus i pass a ref to the engine object..
	void updateNeighborLists(Simulation* simulation, NListDataCollection* nlist_data_collection, volatile bool* finished, int* timing, bool* mutex_lock, const uint32_t step_at_update) {
		auto t0 = std::chrono::high_resolution_clock::now();

		// Make key positions addressable in arrays: compound_key_positions and solvent_positions
		nlist_data_collection->preparePositionData(*simulation, step_at_update);

		// First do culling of neighbors that has left CUTOFF
		NListUtils::cullDistantNeighbors(simulation, nlist_data_collection);


		// Now add compound->solvent, compound->compound
		for (uint16_t id_self = 0; id_self < simulation->n_compounds; id_self++) {

			NeighborList* nlist_self = &nlist_data_collection->compound_neighborlists[id_self];
			HashTable hashtable_compoundneighbors(nlist_self->neighborcompound_ids, nlist_self->n_compound_neighbors, NEIGHBORLIST_MAX_COMPOUNDS * 2);
			HashTable hashtable_solventneighbors(nlist_self->neighborsolvent_ids, nlist_self->n_solvent_neighbors, NEIGHBORLIST_MAX_SOLVENTS * 2);
			const float cutoff_add_self = simulation->compounds_host[id_self].confining_particle_sphere;
			const Float3& pos_self = nlist_data_collection->compound_key_positions[id_self];


			// Go through all solvents in box!
			for (uint16_t id_candidate = 0; id_candidate < simulation->n_solvents; id_candidate++) {
				NeighborList* nlist_candidate = &nlist_data_collection->solvent_neighborlists[id_candidate];
				const Float3& pos_other = nlist_data_collection->solvent_positions[id_candidate];

				addNeighborIfEligible(hashtable_solventneighbors, *nlist_self, *nlist_candidate,
					pos_self, pos_other,
					id_self, id_candidate,
					NeighborList::NEIGHBOR_TYPE::COMPOUND, NeighborList::NEIGHBOR_TYPE::SOLVENT,
					cutoff_add_self
				);
			}

			// Go through all compounds in box, with higher ID than self!
			for (uint16_t id_other = id_self + 1; id_other < simulation->n_compounds; id_other++) {	// For finding new nearby compounds, it is faster and simpler to just check all compounds, since there are so few
				NeighborList* nlist_candidate = &nlist_data_collection->compound_neighborlists[id_other];
				const Float3& pos_other = nlist_data_collection->compound_key_positions[id_other];
				const float cutoff_add_candidate = simulation->compounds_host[id_self].confining_particle_sphere;	// THIS IS BORKEN SINCE LIMAMETRES

				addNeighborIfEligible(hashtable_compoundneighbors, *nlist_self, *nlist_candidate,
					pos_self, pos_other,
					id_self, id_other,
					NeighborList::NEIGHBOR_TYPE::COMPOUND, NeighborList::NEIGHBOR_TYPE::COMPOUND,
					cutoff_add_self + cutoff_add_candidate
				);
			}
		}

		// Finally add all solvent->solvent candidates
		SolventBlockCollection solventblock_collection(nlist_data_collection->solvent_positions, simulation->n_solvents);
		const auto& neighborCandidatesAll = solventblock_collection.getNeighborSolventForAllSolvents(simulation->n_solvents);
		auto t2 = std::chrono::high_resolution_clock::now();

		for (int id_self = 0; id_self < simulation->n_solvents; id_self++) {
			NeighborList& nlist_self = nlist_data_collection->solvent_neighborlists[id_self];
			HashTable hashtable_solventneighbors(nlist_self.neighborsolvent_ids, (int)nlist_self.n_solvent_neighbors, NEIGHBORLIST_MAX_SOLVENTS * 2);
			const Float3& pos_self = nlist_data_collection->solvent_positions[id_self];
			const auto& neighborCandidates = neighborCandidatesAll[id_self];

			for (int i = 0; i < neighborCandidates.n_candidates; i++) {
				const auto id_other = neighborCandidates.candidates[i];
				NeighborList& nlist_candidate = nlist_data_collection->solvent_neighborlists[id_other];
				const Float3 pos_other = nlist_data_collection->solvent_positions[id_other];
				addNeighborIfEligible(hashtable_solventneighbors, nlist_self, nlist_candidate,
					pos_self, pos_other,
					id_self, id_other,
					NeighborList::NEIGHBOR_TYPE::SOLVENT, NeighborList::NEIGHBOR_TYPE::SOLVENT,
					0.f
				);
			}
		}




		Int3 after(nlist_data_collection->compound_neighborlists[0].n_compound_neighbors, nlist_data_collection->compound_neighborlists[0].n_solvent_neighbors, 0);
		//Int3 after(nlist_data_collection->solvent_neighborlists[193].n_compound_neighbors, nlist_data_collection->solvent_neighborlists[193].n_solvent_neighbors, 0);

		//printf("\nEntity went from %d %d neighbors to %d %d\n", before.x, before.y, after.x, after.y);

		auto t1 = std::chrono::high_resolution_clock::now();
		*timing = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();

		//printf("\nSetup time: %d, nlist time: %d\n",
		//	(int)std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t0).count(),
		//	(int)std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t2).count());




		// SIGNALING MAIN THREAD //
		*finished = 1;		// Thread terminates here!
		*mutex_lock = 0;	// Unlock
	}

}

SolventBlockCollection::SolventBlockCollection(const Float3* positions, int n_solvents) {
	for (auto i = 0; i < n_solvents; i++) {
		addSolventId(i, positions[i]);
	}
}

void SolventBlockCollection::addSolventId(uint32_t id, const Float3& pos) {
	const Int3 block_index = getSolventblockIndex(pos);
	m_blocks[block_index.x][block_index.y][block_index.z].insert(id);
}

std::vector<CandidateList> SolventBlockCollection::getNeighborSolventForAllSolvents(uint32_t n_solvents)
{
	std::vector<CandidateList> neighborCandidates(n_solvents);

	const auto& all_indices = getAllIndices();

	//std::for_each(std::execution::par_unseq, all_indices.begin(), all_indices.end(),
	//	[&](const Int3& index) {addAllInsideBlock(neighborCandidates, getBlock(index)); }
	//);

	for (const auto& index : all_indices) {
		// First add all index combinations inside the block
		SolventBlock& block_self = getBlock(index);
		addAllInsideBlock(neighborCandidates, block_self);
		//continue;
		const auto& query_blockindices = getAdjacentIndicesThatAreGreater(index);
		//const auto& query_blockindices = precalcedGreaterIndices[index.x][index.y][index.z];
		for (const auto& query_index : query_blockindices) {
			SolventBlock& block_query = getBlock(query_index);

			// Now add all index combinations between blocks
			addAllBetweenBlocks(neighborCandidates, block_self, block_query);
		}
	}
	return neighborCandidates;
}







Int3 SolventBlockCollection::getSolventblockIndex(const Float3& pos) {
	Float3 hyperpos = pos;
	EngineUtils::applyPBC(&hyperpos);
	return Int3(
		static_cast<int>(hyperpos.x / block_len),
		static_cast<int>(hyperpos.y / block_len),
		static_cast<int>(hyperpos.z / block_len)
	);
}

constexpr std::array<Int3, SolventBlockCollection::blocks_total> SolventBlockCollection::getAllIndices()
{
	std::array<Int3, blocks_total> indices{};
	int index1d = 0;
	for (int z = 0; z < blocks_per_dim; z++) {
		for (int y = 0; y < blocks_per_dim; y++) {
			for (int x = 0; x < blocks_per_dim; x++) {
				indices[index1d++] = { Int3{x, y, z} };
			}
		}
	}
	return indices;
}

constexpr std::array<Int3, 2*2*2-1> SolventBlockCollection::getAdjacentIndicesThatAreGreater(Int3 index)
{
	std::array<Int3, 2*2*2 - 1> indices{};
	int index1d = 0;
	for (int z = index.z; z <= index.z + 1; z++) {
		for (int y = index.y; y <= index.y + 1; y++) {
			for (int x = index.x; x <= index.x + 1; x++) {
				if (x == index.x && y == index.y && z == index.z) { continue; }
				indices[index1d++] = Int3{
					x < blocks_per_dim ? x : 0, 
					y < blocks_per_dim ? y : 0,
					z < blocks_per_dim ? z : 0
				};
			}
		}
	}
	return indices;
}

constexpr std::array<std::array<std::array<std::array<Int3, 2 * 2 * 2 - 1>, SolventBlockCollection::blocks_per_dim>, SolventBlockCollection::blocks_per_dim>, SolventBlockCollection::blocks_per_dim> SolventBlockCollection::precalcGreaterIndices() {
	std::array<std::array<std::array<std::array<Int3, 2 * 2 * 2 - 1>, blocks_per_dim>, blocks_per_dim>, blocks_per_dim> precalcedGreaterIndices;
	for (int z = 0; z < blocks_per_dim; z++) {
		for (int y = 0; y < blocks_per_dim; y++) {
			for (int x = 0; x < blocks_per_dim; x++) {
				precalcedGreaterIndices[x][y][z] = getAdjacentIndicesThatAreGreater(Int3{ x, y, z });
			}
		}
	}
	return precalcedGreaterIndices;
}

SolventBlockCollection::SolventBlock& SolventBlockCollection::getBlock(const Int3& index) {
	return m_blocks[index.x][index.y][index.z];
}

void SolventBlockCollection::addAllInsideBlock(std::vector<CandidateList>& neighborCandidates, const SolventBlock& block) {
	for (auto i = 0; i < block.n_elements; i++) {
		for (int ii = i+1; ii < block.n_elements; ii++) {
			const auto& id_a = block.solvent_ids[i];
			const auto& id_b = block.solvent_ids[ii];
			if (id_a == id_b) {
				int bb;
				exit(1);
			}
			neighborCandidates[id_a].candidates[neighborCandidates[id_a].n_candidates++] = id_b;
			neighborCandidates[id_b].candidates[neighborCandidates[id_b].n_candidates++] = id_a;
		}
	}
}

void SolventBlockCollection::addAllBetweenBlocks(std::vector<CandidateList>& neighborCandidates, const SolventBlock& blocka, const SolventBlock& blockb) {
	for (auto ia = 0; ia < blocka.n_elements; ia++) {
		for (int ib = 0; ib < blockb.n_elements; ib++) {
			const auto& id_a = blocka.solvent_ids[ia];
			const auto& id_b = blockb.solvent_ids[ib];
			if (id_a == id_b) {
				int bb;
				exit(1);
			}
			neighborCandidates[id_a].candidates[neighborCandidates[id_a].n_candidates++] = id_b;
			neighborCandidates[id_b].candidates[neighborCandidates[id_b].n_candidates++] = id_a;
		}
	}
}