#include "Neighborlists.cuh"


NListManager::NListManager(Simulation* simulation) {
	nlist_data_collection = new NListDataCollection(simulation);


	for (int i = 0; i < nlist_data_collection->n_compounds; i++) {
		nlist_data_collection->compound_neighborlists[i].associated_id = i;
	}
	for (int i = 0; i < nlist_data_collection->n_solvents; i++) {
		nlist_data_collection->solvent_neighborlists[i].associated_id = i;
	}
}


void NListManager::updateNeighborLists(Simulation* simulation, bool* updatenlists_mutexlock, bool force_update, bool async, int* timings) {
	if (async && !force_update) {
		std::thread nlist_worker(NListUtils::updateNeighborLists, simulation, nlist_data_collection, &updated_neighborlists_ready, timings, updatenlists_mutexlock);
		nlist_worker.detach();
	}
	else {
		NListUtils::updateNeighborLists(simulation, nlist_data_collection, &updated_neighborlists_ready, timings, updatenlists_mutexlock);
	}

	prev_update_step = simulation->getStep();

	if (force_update) {
		Int3 n_data(nlist_data_collection->compound_neighborlists[0].n_compound_neighbors, nlist_data_collection->compound_neighborlists[0].n_solvent_neighbors, 0);
		printf("\nEntity neighbors: %d %d\n", n_data.x, n_data.y);
	}
}

void NListManager::offloadPositionDataNLIST(Simulation* simulation) {
//	if (simulation->n_solvents > 0)
//		hipMemcpy(nlist_data_collection->solvents, simulation->box->solvents, sizeof(Solvent) * simulation->n_solvents, hipMemcpyDeviceToHost);
}

void NListManager::pushNlistsToDevice(Simulation* simulation) {
	hipMemcpy(simulation->box->compound_neighborlists, nlist_data_collection->compound_neighborlists, sizeof(NeighborList) * simulation->n_compounds, hipMemcpyHostToDevice);
	hipMemcpy(simulation->box->solvent_neighborlists, nlist_data_collection->solvent_neighborlists, sizeof(NeighborList) * simulation->n_solvents, hipMemcpyHostToDevice);
	updated_neighborlists_ready = 0;
}



namespace NListUtils {

	bool neighborWithinCutoff(const Float3* pos_a, const Float3* pos_b, const float cutoff_lm) {		// This is used for compounds with a confining_particle_sphere from key_particle BEFORE CUTOFF begins
		const float dist = EngineUtils::calcHyperDist(pos_a, pos_b);		
		return dist < cutoff_lm;
	}


	void cullDistantNeighbors(Simulation* simulation, NListDataCollection* nlist_data_collection) {
		for (int id_self = 0; id_self < nlist_data_collection->n_compounds; id_self++) {
			NeighborList* nlist_self = &nlist_data_collection->compound_neighborlists[id_self];
			float cutoff_add_self = simulation->compounds_host[id_self].confining_particle_sphere;



			for (int j = 0; j < nlist_self->n_compound_neighbors; j++) {		// Cull compound-compound
				int id_neighbor = nlist_self->neighborcompound_ids[j];
				NeighborList* nlist_neighbor = &nlist_data_collection->compound_neighborlists[id_neighbor];
				float cutoff_add_neighbor = simulation->compounds_host[id_neighbor].confining_particle_sphere;

				if (id_self < id_neighbor) {
					if (!neighborWithinCutoff(&nlist_data_collection->compound_key_positions[id_self], &nlist_data_collection->compound_key_positions[id_neighbor], cutoff_add_self + cutoff_add_neighbor + CUTOFF_LM)) {
						nlist_self->removeId(id_neighbor, NeighborList::NEIGHBOR_TYPE::COMPOUND);
						nlist_neighbor->removeId(id_self, NeighborList::NEIGHBOR_TYPE::COMPOUND);
						j--;	// Decrement, as the removeId puts the last element at the current and now vacant spot.
					}
				}
			}


			for (int j = 0; j < nlist_self->n_solvent_neighbors; j++) {			// Cull compound-solvent
				int id_neighbor = nlist_self->neighborsolvent_ids[j];
				NeighborList* nlist_neighbor = &nlist_data_collection->solvent_neighborlists[id_neighbor];

				//printf("Dist: %f\n", (nlist_data_collection->compound_key_positions[id_self] - nlist_data_collection->solvent_positions[id_neighbor]).len());
				if (!neighborWithinCutoff(&nlist_data_collection->compound_key_positions[id_self], &nlist_data_collection->solvent_positions[id_neighbor], cutoff_add_self + CUTOFF_LM) && false) {
					nlist_self->removeId(id_neighbor, NeighborList::NEIGHBOR_TYPE::SOLVENT);
					//	printf("J: %d\n", j);
					nlist_neighbor->removeId(id_self, NeighborList::NEIGHBOR_TYPE::COMPOUND);
					j--;	// Decrement, as the removeId puts the last element at the current and now va-cant spot.
				}
			}
		}


		for (int id_self = 0; id_self < nlist_data_collection->n_solvents; id_self++) {																// Cull solvent-solvent
			NeighborList* nlist_self = &nlist_data_collection->solvent_neighborlists[id_self];

			int cnt = 0;

			for (int j = 0; j < nlist_self->n_solvent_neighbors; j++) {			/// NOT FINISHED HERE
				int id_neighbor = nlist_self->neighborsolvent_ids[j];
				NeighborList* nlist_neighbor = &nlist_data_collection->solvent_neighborlists[id_neighbor];

				if (!neighborWithinCutoff(&nlist_data_collection->solvent_positions[id_self], &nlist_data_collection->solvent_positions[id_neighbor], CUTOFF_LM)) {
					cnt++;
					if (!nlist_self->removeId(id_neighbor, NeighborList::NEIGHBOR_TYPE::SOLVENT))
						printf("J1: %d id_self %d id_neighbor %d    cnt %d\n", j, id_self, id_neighbor, cnt);
					if (!nlist_neighbor->removeId(id_self, NeighborList::NEIGHBOR_TYPE::SOLVENT)) {
						printf("J2: %d of %d.   id_self %d id_neighbor %d count: %d\n", j, nlist_self->n_solvent_neighbors, id_self, id_neighbor, cnt);
						for (int i = 0; i < nlist_self->n_solvent_neighbors; i++) {
							printf("neighbor %d\n", nlist_self->neighborsolvent_ids[i]);
						}
						printf("\n\n\n");
						exit(1);
					}


					j--;	// Decrement, as the removeId puts the last element at the current and now vacant spot.
				}
			}
		}
	}




	void inline addNeighborIfEligible(HashTable& currentNeighbors, 
		NeighborList& nlist_self, NeighborList& nlist_other, 
		const Float3& pos_self, const Float3& pos_other, 
		const int& id_self, const int& id_other, 
		NeighborList::NEIGHBOR_TYPE neighbortype_self, NeighborList::NEIGHBOR_TYPE neighbortype_other,
		const float cutoff_extension) 
	{
		if (neighborWithinCutoff(&pos_self, &pos_other, CUTOFF_LM + cutoff_extension)) {
			if (currentNeighbors.insert(id_other)) {
				nlist_self.addId(id_other, neighbortype_other);
				nlist_other.addId(id_self, neighbortype_self);
			}
		}
	}


	void updateNeighborLists(Simulation* simulation, NListDataCollection* nlist_data_collection, volatile bool* finished, int* timing, bool* mutex_lock) {	// This is a thread worker-function, so it can't own the object, thus i pass a ref to the engine object..
		auto t0 = std::chrono::high_resolution_clock::now();
		//Int3 before(nlist_data_collection->compound_neighborlists[0].n_compound_neighbors, nlist_data_collection->compound_neighborlists[0].n_solvent_neighbors, 0);

		// Make key positions addressable in arrays: compound_key_positions and solvent_positions
		//nlist_data_collection->preparePositionData(simulation->compounds_host);		
		nlist_data_collection->preparePositionData(*simulation);

		// First do culling of neighbors that has left CUTOFF
		cullDistantNeighbors(simulation, nlist_data_collection);


		// Now add compound->solvent, compound->compound
		for (uint16_t id_self = 0; id_self < simulation->n_compounds; id_self++) {

			NeighborList* nlist_self = &nlist_data_collection->compound_neighborlists[id_self];
			HashTable hashtable_compoundneighbors(nlist_self->neighborcompound_ids, nlist_self->n_compound_neighbors, NEIGHBORLIST_MAX_COMPOUNDS * 2);
			HashTable hashtable_solventneighbors(nlist_self->neighborsolvent_ids, nlist_self->n_solvent_neighbors, NEIGHBORLIST_MAX_SOLVENTS * 2);
			const float cutoff_add_self = simulation->compounds_host[id_self].confining_particle_sphere;
			const Float3& pos_self = nlist_data_collection->compound_key_positions[id_self];


			// Go through all solvents in box!
			for (uint16_t id_candidate = 0; id_candidate < simulation->n_solvents; id_candidate++) {
				NeighborList* nlist_candidate = &nlist_data_collection->solvent_neighborlists[id_candidate];
				const Float3& pos_other = nlist_data_collection->solvent_positions[id_candidate];

				addNeighborIfEligible(hashtable_solventneighbors, *nlist_self, *nlist_candidate,
					pos_self, pos_other,
					id_self, id_candidate,
					NeighborList::NEIGHBOR_TYPE::COMPOUND, NeighborList::NEIGHBOR_TYPE::SOLVENT,
					cutoff_add_self
				);
			}

			// Go through all compounds in box, with higher ID than self!
			for (uint16_t id_other = id_self + 1; id_other < simulation->n_compounds; id_other++) {	// For finding new nearby compounds, it is faster and simpler to just check all compounds, since there are so few
				NeighborList* nlist_candidate = &nlist_data_collection->compound_neighborlists[id_other];
				const Float3& pos_other = nlist_data_collection->compound_key_positions[id_other];
				float cutoff_add_candidate = simulation->compounds_host[id_self].confining_particle_sphere;	// THIS IS BORKEN SINCE LIMAMETRES

				addNeighborIfEligible(hashtable_compoundneighbors, *nlist_self, *nlist_candidate,
					pos_self, pos_other,
					id_self, id_other,
					NeighborList::NEIGHBOR_TYPE::COMPOUND, NeighborList::NEIGHBOR_TYPE::COMPOUND,
					cutoff_add_self + cutoff_add_candidate
				);
			}
		}

		// Finally add all solvent->solvent candidates
		for (int id_self = 0; id_self < simulation->n_solvents; id_self++) {
			NeighborList* nlist_self = &nlist_data_collection->solvent_neighborlists[id_self];
			HashTable hashtable_solventneighbors(nlist_self->neighborsolvent_ids, (int)nlist_self->n_solvent_neighbors, NEIGHBORLIST_MAX_SOLVENTS * 2);
			const Float3& pos_self = nlist_data_collection->solvent_positions[id_self];


			for (int id_other = id_self + 1; id_other < simulation->n_solvents; id_other++) {
				NeighborList* nlist_candidate = &nlist_data_collection->solvent_neighborlists[id_other];
				const Float3 pos_other = nlist_data_collection->solvent_positions[id_other];

				addNeighborIfEligible(hashtable_solventneighbors, *nlist_self, *nlist_candidate,
					pos_self, pos_other,
					id_self, id_other,
					NeighborList::NEIGHBOR_TYPE::SOLVENT, NeighborList::NEIGHBOR_TYPE::SOLVENT,
					0.f
				);
			}
		}

		Int3 after(nlist_data_collection->compound_neighborlists[0].n_compound_neighbors, nlist_data_collection->compound_neighborlists[0].n_solvent_neighbors, 0);
		//Int3 after(nlist_data_collection->solvent_neighborlists[193].n_compound_neighbors, nlist_data_collection->solvent_neighborlists[193].n_solvent_neighbors, 0);

		//printf("\nEntity went from %d %d neighbors to %d %d\n", before.x, before.y, after.x, after.y);

		auto t1 = std::chrono::high_resolution_clock::now();
		*timing = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();




		// SIGNALING MAIN THREAD //
		*finished = 1;		// Thread terminates here!
		*mutex_lock = 0;	// Unlock
	}
}
