#include "hip/hip_runtime.h"
#include "Analyzer.cuh"
#include "Printer.h"
#include "PhysicsUtils.cuh"
#include "DeviceAlgorithms.cuh"
#include "Constants.h"
#include "Filehandling.h"



using namespace LIMA_Print;

const int THREADS_PER_SOLVENTBLOCK_ANALYZER = 128;




void __global__ monitorCompoundEnergyKernel(Compound* compounds, const ForceField_NB* const forcefield, const BoxParams boxparams, float* potE_buffer, float* vel_buffer, Float3* data_out) {		// everything here breaks if not all compounds are identical in particle count and particle mass!!!!!!!
	__shared__ Float3 energy[MAX_COMPOUND_PARTICLES];
	__shared__ Compound compound;


	const int64_t step = blockIdx.x;	// Step relative to current batch
	const int64_t compound_index = blockIdx.y;
	const int64_t particle_index = threadIdx.x;
	energy[particle_index] = Float3(0.f);


	if (particle_index == 0) {
		data_out[compound_index + (step) * boxparams.n_compounds] = Float3{};
		compound = compounds[compound_index];
	}
	__syncthreads();

	if (particle_index >= compound.n_particles) {
		return;
	}
	__syncthreads();

	const uint8_t atom_type = compound.atom_types[particle_index];
	const float mass = forcefield->particle_parameters[atom_type].mass;

	const int64_t compound_offset = compound_index * MAX_COMPOUND_PARTICLES;
	const int64_t step_offset = step * boxparams.total_particles_upperbound;
	const float potE = potE_buffer[particle_index + compound_offset + step_offset];

	const float speed = vel_buffer[particle_index + compound_offset + step_offset];
	const float kinE = PhysicsUtils::calcKineticEnergy(speed, mass);	// remove direction from vel

	const float totalE = potE + kinE;

	energy[particle_index] = Float3(potE, kinE, totalE);
	__syncthreads();

	LAL::distributedSummation(energy, MAX_COMPOUND_PARTICLES);
	__syncthreads();

	if (particle_index == 0) {
		data_out[compound_index + (step) * boxparams.n_compounds] = energy[0];
	}
}





void __global__ monitorSolventEnergyKernel(const BoxParams boxparams, float* potE_buffer, float* vel_buffer, Float3* data_out) {
	__shared__ Float3 energy[THREADS_PER_SOLVENTBLOCK_ANALYZER];


	const int solvent_index = threadIdx.x + blockIdx.y * THREADS_PER_SOLVENTBLOCK_ANALYZER;
	const int step = blockIdx.x;
	const int compounds_offset = boxparams.n_compounds * MAX_COMPOUND_PARTICLES;
	const int step_offset = step * boxparams.total_particles_upperbound;

	energy[threadIdx.x] = Float3(0.f);
	if (threadIdx.x == 0) {
		data_out[(step) * gridDim.y + blockIdx.y] = energy[0];
	}
	if (solvent_index >= boxparams.n_solvents) { return; }


	const float velocity = vel_buffer[step_offset + compounds_offset + solvent_index];
	const float kinE = PhysicsUtils::calcKineticEnergy(velocity, SOLVENT_MASS);	// remove direction from vel
	float potE = potE_buffer[compounds_offset + solvent_index + step * boxparams.total_particles_upperbound];

	const float totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();
	LAL::distributedSummation(energy, THREADS_PER_SOLVENTBLOCK_ANALYZER);
	if (threadIdx.x == 0) {
		data_out[(step) * gridDim.y + blockIdx.y] = energy[0];
	}
}


const int NUM_BINS = 16;
const int BIN_BASE = 10;
__device__ int getBinIndex(float value) {
	int binIndex = (value > 0) ? log10f(value) / log10f(BIN_BASE) : -(log10f(-value) / log10f(BIN_BASE));
	binIndex += NUM_BINS / 2;  // Center the bins around zero
	return std::min(std::max(binIndex, 0), NUM_BINS - 1);  // Clamp to valid range
}

__global__ void potEHistogramKernel(Compound* compounds, int total_particles_upperbound, float* potE_buffer, int* histogramData, int step) {
	__shared__ int shared_histogram[NUM_BINS];

	const int compound_index = blockIdx.x;   // Unique index for each compound
	const int particle_index = threadIdx.x;  // Unique index for each particle within a compound

	// Initialize shared histogram to zero
	if (particle_index < NUM_BINS) {
		shared_histogram[particle_index] = 0;
	}
	__syncthreads();
	if (particle_index >= compounds[compound_index].n_particles) {
		return;
	}

	// Calculate the offsets
	const int64_t compound_offset = compound_index * MAX_COMPOUND_PARTICLES;
	const float potE = potE_buffer[particle_index + compound_offset];
	// Determine the bin index for the current potential energy
	int binIndex = getBinIndex(potE);

	// Atomically increment the appropriate bin in the shared histogram
	atomicAdd(&shared_histogram[binIndex], 1);
	__syncthreads();

	// First thread writes the shared histogram to the global histogram
	if (particle_index == 0) {
		for (int i = 0; i < NUM_BINS; ++i) {
			if (shared_histogram[i] > 0) {
				atomicAdd(&histogramData[i], shared_histogram[i]);
			}
		}
	}
}










Analyzer::AnalyzedPackage Analyzer::analyzeEnergy(Simulation* simulation) {	// Calculates the avg J/mol // calculate energies separately for compounds and solvents. weigh averages based on amount of each
	LIMA_UTILS::genericErrorCheck("Cuda error before analyzeEnergy\n");

	const int64_t n_entryindices = LIMALOGSYSTEM::getMostRecentDataentryIndex(simulation->getStep(), simulation->simparams_host.data_logging_interval);

	if (n_entryindices < 2) { return Analyzer::AnalyzedPackage(); }


	// First set up some stuff needed on device, that is currently on host
	hipMalloc(&forcefield_device, sizeof(ForceField_NB));
	hipMemcpy(forcefield_device, &simulation->forcefield, sizeof(ForceField_NB), hipMemcpyHostToDevice);

	if (simulation->boxparams_host.n_compounds > 0) {
		hipMalloc(&compounds_device, sizeof(Compound) * simulation->compounds_host.size());
		hipMemcpy(compounds_device, simulation->compounds_host.data(), sizeof(Compound) * simulation->compounds_host.size(), hipMemcpyHostToDevice);
	}
	


	std::vector<Float3> average_energy;
	average_energy.resize(n_entryindices - 2);	// Ignore first and last step	// TODO: Rework this, no longer necessary as we use VVS

	// We need to split up the analyser into steps, as we cannot store all positions traj on device at once.
	int64_t max_steps_per_kernel = 100;
	int64_t particles_per_step = simulation->boxparams_host.total_particles_upperbound;
	int64_t max_values_per_kernel = max_steps_per_kernel * particles_per_step;							// Pad steps with 2 for vel calculation

	const std::string bytesize = std::to_string((sizeof(Float3) + sizeof(double)) * (max_values_per_kernel) * 1e-6);
	m_logger->print("Analyzer malloc " + bytesize + " MB on device\n");
	hipMalloc(&potE_buffer_device, sizeof(float) * max_values_per_kernel);
	hipMalloc(&vel_buffer_device, sizeof(float) * max_values_per_kernel);

	for (int64_t i = 0; i < ceil((double)n_entryindices / (double)max_steps_per_kernel); i++) {
		const int64_t step_offset = i * max_steps_per_kernel;												// offset one since we can't analyse step 1
		const int64_t steps_in_kernel = std::min(max_steps_per_kernel, n_entryindices - step_offset);

		hipMemcpy(potE_buffer_device, &simulation->potE_buffer->data()[step_offset * particles_per_step], sizeof(float) * steps_in_kernel * particles_per_step, hipMemcpyHostToDevice);
		hipMemcpy(vel_buffer_device, &simulation->vel_buffer->data()[step_offset * particles_per_step], sizeof(float) * steps_in_kernel * particles_per_step, hipMemcpyHostToDevice);
		LIMA_UTILS::genericErrorCheck("Cuda error during analyzer transfer2\n");

		std::vector<Float3> average_solvent_energy = analyzeSolvateEnergy(simulation, steps_in_kernel);
		std::vector<Float3> average_compound_energy = analyzeCompoundEnergy(simulation, steps_in_kernel);

		for (int64_t ii = 0; ii < steps_in_kernel; ii++) {
			int64_t step = step_offset + ii - 1;	// -1 because index 0 is unused
			if (step == -1 || step >= n_entryindices -2u) { continue; }	// Dont save first step, as the kinE is slightly wrong
			average_energy[step] = (average_solvent_energy[ii] + average_compound_energy[ii]);
		}
	}

	hipFree(potE_buffer_device);
	hipFree(vel_buffer_device);
	hipFree(forcefield_device);
	if (simulation->boxparams_host.n_compounds > 0) {
		hipFree(compounds_device);
	}

	m_logger->finishSection("Finished analyzing energies");
	return AnalyzedPackage(average_energy, simulation->temperature_buffer);
}

std::vector<Float3> Analyzer::analyzeSolvateEnergy(Simulation* simulation, uint64_t n_steps) {
	// Start by creating array of energies of value 0
	std::vector<Float3> average_solvent_energy(n_steps);

	int blocks_per_solventkernel = (int)ceil((float)simulation->boxparams_host.n_solvents / (float)THREADS_PER_SOLVENTBLOCK_ANALYZER);

	// If any solvents are present, fill above array
	if (simulation->boxparams_host.n_solvents > 0) {

		std::vector<Float3> average_solvent_energy_blocked(n_steps * blocks_per_solventkernel);
		Float3* data_out;
		hipMalloc(&data_out, sizeof(Float3) * blocks_per_solventkernel * n_steps);

		dim3 block_dim(n_steps, blocks_per_solventkernel, 1);
		monitorSolventEnergyKernel << < block_dim, THREADS_PER_SOLVENTBLOCK_ANALYZER >> > (simulation->boxparams_host, potE_buffer_device, vel_buffer_device, data_out);	// TODO: FIx
		LIMA_UTILS::genericErrorCheck("Cuda error during analyzeSolvateEnergy\n");

		hipMemcpy(average_solvent_energy_blocked.data(), data_out, sizeof(Float3) * blocks_per_solventkernel * n_steps, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(data_out);

		for (uint64_t step = 0; step < n_steps; step++) {
			average_solvent_energy[step] = Float3(0.f);
			for (int block = 0; block < blocks_per_solventkernel; block++) {
				average_solvent_energy[step] += average_solvent_energy_blocked[block + step * blocks_per_solventkernel];
			}
			//average_solvent_energy[step] *= (1.f / simulation->boxparams_host.n_solvents);
		}

	}

	return average_solvent_energy;
}


std::vector<Float3> Analyzer::analyzeCompoundEnergy(Simulation* simulation, uint64_t steps_in_kernel) {
	const uint64_t n_datapoints = simulation->boxparams_host.n_compounds * steps_in_kernel;

	std::vector<Float3> total_compound_energy(steps_in_kernel);

	if (simulation->extraparams.total_compound_particles > 0) {
		std::vector<Float3> host_data(n_datapoints);

		Float3* data_out;
		hipMalloc(&data_out, sizeof(Float3) * n_datapoints);

		dim3 block_dim(static_cast<uint32_t>(steps_in_kernel), simulation->boxparams_host.n_compounds, 1);
		monitorCompoundEnergyKernel << < block_dim, MAX_COMPOUND_PARTICLES >> > (compounds_device, forcefield_device, simulation->boxparams_host, potE_buffer_device, vel_buffer_device, data_out);
		hipDeviceSynchronize();
		LIMA_UTILS::genericErrorCheck("Cuda error during analyzeCompoundEnergy\n");

		hipMemcpy(host_data.data(), data_out, sizeof(Float3) * n_datapoints, hipMemcpyDeviceToHost);
		hipFree(data_out);


		for (uint64_t step = 0; step < steps_in_kernel; step++) {
			for (uint64_t i = 0; i < simulation->boxparams_host.n_compounds; i++) {
				total_compound_energy[step] += host_data[i + step * simulation->boxparams_host.n_compounds];
			}
		}

	}

	return total_compound_energy;
}

float getMin(const std::vector<float>& vec) {
	return *std::min_element(vec.begin(), vec.end());
}

float getMax(const std::vector<float>& vec) {
	return *std::max_element(vec.begin(), vec.end());
}

float getVarianceCoefficient(const std::vector<float>& vec) {
	if (vec.empty()) { return 0.f; } 
	const float stddev = getStdDev(vec);
	const float mean = getMean(vec);

	if (stddev == 0.f && mean == 0.f) { return 0.f; }
	return  stddev / std::abs(mean);
}

void printRow(string title, std::vector<float>& vec) {
	if (vec.empty()) { return; }
	LIMA_Printer::printTableRow(
		title, { 
			getMin(vec), 
			getMax(vec), 
			getStdDev(vec),
			(vec.back() - vec.front()) / vec.front() });
}

void Analyzer::printEnergy(AnalyzedPackage* package) {
	LIMA_Printer::printTableRow({ "", "min", "max", "Std. deviation", "Change 0->n"});
	printRow("potE", package->pot_energy);
	printRow("kinE", package->kin_energy);
	printRow("totalE", package->total_energy);
}





float calculateSlopeLinearRegression(const std::vector<float>& y_values, const float mean) {
	size_t n = y_values.size();
	float sum_x = 0;
	float sum_y = 0;
	float sum_xy = 0;
	float sum_xx = 0;

	for (size_t i = 0; i < n; ++i) {
		sum_x += i;
		sum_y += y_values[i];
		sum_xy += i * y_values[i];
		sum_xx += i * i;
	}

	const float slope = (n * sum_xy - sum_x * sum_y) / (n * sum_xx - sum_x * sum_x);
	const float slope_coefficient = slope / mean;
	return slope_coefficient;
}

Analyzer::AnalyzedPackage::AnalyzedPackage(std::vector<Float3>& avg_energy, std::vector<float> temperature) {
	energy_data = avg_energy;
	//auto e_cnt = energy_data.size();

	temperature_data = temperature;
	//memcpy(temperature_data.data(), t_ptr, t_cnt);

	auto e_cnt = energy_data.size();
	pot_energy.resize(e_cnt);
	kin_energy.resize(e_cnt);
	total_energy.resize(e_cnt);
	for (int i = 0; i < e_cnt; i++) {
		pot_energy[i] = energy_data[i].x;
		kin_energy[i] = energy_data[i].y;
		total_energy[i] = energy_data[i].z;
	}

	mean_energy = getMean(total_energy);

	energy_gradient = calculateSlopeLinearRegression(total_energy, mean_energy);
	variance_coefficient = getVarianceCoefficient(total_energy);
}


























void Analyzer::findAndDumpPiecewiseEnergies(const Simulation& sim, const std::string& workdir) {
	std::vector<float> energies;
	
	for (auto entryindex = 0; entryindex < LIMALOGSYSTEM::getMostRecentDataentryIndex(sim.getStep()-1, sim.simparams_host.data_logging_interval); entryindex++) {

		for (int compound_id = 0; compound_id < sim.boxparams_host.n_compounds; compound_id++) {
			for (int particle_id = 0; particle_id < MAX_COMPOUND_PARTICLES; particle_id++) {
				
				const float potE = sim.potE_buffer->getCompoundparticleDatapointAtIndex(compound_id, particle_id, entryindex);

				const uint8_t& atom_type = sim.compounds_host[compound_id].atom_types[particle_id];
				const float mass = sim.forcefield.particle_parameters[atom_type].mass;
				const float vel = sim.vel_buffer->getCompoundparticleDatapointAtIndex(compound_id, particle_id, entryindex);
				const float kinE = PhysicsUtils::calcKineticEnergy(vel, mass);
				
				energies.emplace_back(potE);
				energies.emplace_back(kinE);
			}
		}

		for (int solvent_id = 0; solvent_id < sim.boxparams_host.n_solvents; solvent_id++) {

			const float potE = sim.potE_buffer->getSolventparticleDatapointAtIndex(solvent_id, entryindex);

			const float mass = sim.forcefield.particle_parameters[ATOMTYPE_SOLVENT].mass;
			const float vel = sim.vel_buffer->getSolventparticleDatapointAtIndex(solvent_id, entryindex);
			const float kinE = PhysicsUtils::calcKineticEnergy(vel, mass);

			energies.emplace_back(potE);
			energies.emplace_back(kinE);
		}
	}

	Filehandler::dumpToFile(energies.data(), energies.size(), workdir + "/PiecewiseEnergy.bin");
}




std::vector<int64_t> MakeBinLabels() {
	std::vector<int64_t> bins;

	int64_t current_bin = 10;

	while (bins.size() < NUM_BINS / 2) {
		bins.push_back(current_bin);
		current_bin = (current_bin == 0) ? 10 : current_bin * 10;
	}

	std::vector<double> negative_bins;
	current_bin = -10;
	while (negative_bins.size() < NUM_BINS / 2) {
		negative_bins.push_back(current_bin);
		current_bin *= 10;
	}

	std::reverse(negative_bins.begin(), negative_bins.end());
	bins.insert(bins.begin(), negative_bins.begin(), negative_bins.end());

	return bins;
}

void SimAnalysis::PlotPotentialEnergyDistribution(const Simulation& simulation, const std::filesystem::path& dir, const std::vector<int>& stepsToPlot) {
	int* histogramDataDevice;
	hipMalloc(&histogramDataDevice, NUM_BINS * sizeof(int));
		
	float* energyBufferDevice;	
	hipMalloc(&energyBufferDevice, sizeof(float) * simulation.boxparams_host.total_particles_upperbound);

	Compound* compoundsDevice;
	hipMalloc(&compoundsDevice, sizeof(Compound) * simulation.compounds_host.size());
	hipMemcpy(compoundsDevice, simulation.compounds_host.data(), sizeof(Compound) * simulation.compounds_host.size(), hipMemcpyHostToDevice);

	std::ofstream out_file(dir / "histogram_data.bin", std::ios::binary);
	int nPlots = stepsToPlot.size();
	out_file.write(reinterpret_cast<char*>(&nPlots), sizeof(int));
	for (int step : stepsToPlot) {
		hipMemcpy(energyBufferDevice, simulation.potE_buffer->GetBufferAtStep(step), sizeof(float) * simulation.boxparams_host.total_particles_upperbound, hipMemcpyHostToDevice);
		hipMemset(histogramDataDevice, 0, NUM_BINS * sizeof(int));

		hipDeviceSynchronize();
		potEHistogramKernel << <simulation.boxparams_host.n_compounds, MAX_COMPOUND_PARTICLES >> > (compoundsDevice, simulation.boxparams_host.total_particles_upperbound, energyBufferDevice, histogramDataDevice, step);
		hipDeviceSynchronize();

		std::vector<int> histogramDataHost;
		GenericCopyToHost(histogramDataDevice, histogramDataHost, NUM_BINS);

		std::vector<int64_t> bins = MakeBinLabels();
		
		out_file.write(reinterpret_cast<char*>(bins.data()), bins.size() * sizeof(int64_t));
		out_file.write(reinterpret_cast<char*>(histogramDataHost.data()), histogramDataHost.size() * sizeof(int));
	}
	out_file.close();

	hipFree(energyBufferDevice);
	hipFree(compoundsDevice);
}