#include "hip/hip_runtime.h"
#include "LIMA_MD/src/Rasterizer.cuh"
#include "LIMA_ENGINE/include/EngineUtils.cuh"
#include "LIMA_BASE/include/Utilities.h"


#include <algorithm>
#include <cuda/std/cmath>



std::vector<RenderBall> Rasterizer::render(Simulation* simulation) {
	solvent_offset = simulation->n_compounds * MAX_COMPOUND_PARTICLES;
	n_threadblocks = (int)ceil((float)simulation->total_particles_upperbound / (float)RAS_THREADS_PER_BLOCK);

    LIMA_UTILS::genericErrorCheck("Error before renderer");
	RenderAtom* atoms_dev = getAllAtoms(simulation);

    std::vector<RenderBall> balls_host = processAtoms(atoms_dev, simulation);
    hipFree(atoms_dev);

    std::sort(balls_host.begin(), balls_host.end(), [](const RenderBall& a, const RenderBall& b) {return !a.disable && a.pos.y < b.pos.y; });    

    LIMA_UTILS::genericErrorCheck("Error after renderer");

	return balls_host;
}












__global__ void loadCompoundatomsKernel(Box* box, RenderAtom* atoms, const int step);
__global__ void loadSolventatomsKernel(Box* box, RenderAtom* atoms, int offset, const int step);
__global__ void processAtomsKernel(RenderAtom* atoms, RenderBall* balls);

__global__ void kernelA(Box* box) {
    // Do nothing
}
__global__ void kernelB(Box* box) {
    // Do nothing
}

RenderAtom* Rasterizer::getAllAtoms(Simulation* simulation) {
	RenderAtom* atoms;
	hipMallocManaged(&atoms, sizeof(RenderAtom) * simulation->total_particles_upperbound);


    //kernelA << <10, 10>> > (simulation->sim_dev->box);
    //hipDeviceSynchronize();
    //kernelB << <10, 10>> > (simulation->sim_dev->box);


    Box* boxptr = simulation->sim_dev->box; // Use intermediate ptr to avoid prepascal limitation of concurrent managed data access
    if (simulation->n_compounds > 0) {
        loadCompoundatomsKernel << <simulation->n_compounds, MAX_COMPOUND_PARTICLES >> > (boxptr, atoms, simulation->simparams_host.step);
    }
	if (simulation->n_solvents > 0) {
		loadSolventatomsKernel << < SolventBlockGrid::blocks_total, MAX_SOLVENTS_IN_BLOCK >> > (boxptr, atoms, solvent_offset, simulation->simparams_host.step);
	}

	return atoms;
}


std::vector<RenderBall> Rasterizer::processAtoms(RenderAtom* atoms, Simulation* simulation) {
    RenderBall* balls_device;
    hipMalloc(&balls_device, sizeof(RenderBall) * simulation->total_particles_upperbound);
    processAtomsKernel <<< n_threadblocks, RAS_THREADS_PER_BLOCK >>> (atoms, balls_device);
    LIMA_UTILS::genericErrorCheck("Error during rendering");

    std::vector<RenderBall> balls_host(simulation->total_particles_upperbound);    
    hipMemcpy(balls_host.data(), balls_device, sizeof(RenderBall) * simulation->total_particles_upperbound, hipMemcpyDeviceToHost);

    hipFree(balls_device);

    return balls_host;
}







__device__ ATOM_TYPE RAS_getTypeFromIndex(int atom_index) {
    switch (atom_index)
    {
    case 0:
        return ATOM_TYPE::SOL;
    case 1:
        return ATOM_TYPE::C;
    case 2:
        return ATOM_TYPE::O;
    case 3:
        return ATOM_TYPE::N;
    case 4:
        return ATOM_TYPE::H;
    case 5: 
        return ATOM_TYPE::P;
    case 6:
        return ATOM_TYPE::SOL;
    default:
        return ATOM_TYPE::NONE;
    }
}

__device__ ATOM_TYPE RAS_getTypeFromMass(double mass) {
    mass *= 1000.f;   //convert to g
    if (mass < 4)
        return ATOM_TYPE::H;
    if (mass < 14)
        return ATOM_TYPE::C;
    if (mass < 15)
        return ATOM_TYPE::N;
    if (mass < 18)
        return ATOM_TYPE::O;
    if (mass < 32)
        return ATOM_TYPE::P;
    return ATOM_TYPE::NONE;
}

__device__ Int3 getColor(ATOM_TYPE atom_type) {
    switch (atom_type)
    {
    case ATOM_TYPE::SOL:
        return Int3(0x03, 0xa9, 0xf4);
    case ATOM_TYPE::H:
        return Int3(0xF1, 0xF1, 0xF1);
    case ATOM_TYPE::O:
        return Int3(0xE0, 0x20, 0x20);
    case ATOM_TYPE::C:
        return Int3(0x30, 0x10, 0x90);
    case ATOM_TYPE::P:
        return Int3(0xFC, 0xF7, 0x5E);
    case ATOM_TYPE::N:
        return Int3(0x2E, 0x8B, 0x57);
    case ATOM_TYPE::NONE:
        return Int3(0xF2, 0xE5, 0xD9);     
    default:
        return Int3(0, 0, 0);
    }
}

__device__ float getRadius(ATOM_TYPE atom_type) {
    switch (atom_type)
    {

    case ATOM_TYPE::H:
        return 0.04;
    case ATOM_TYPE::C:
        return 0.1;
    case ATOM_TYPE::N:
        return 0.1;
    case ATOM_TYPE::O:
        return 0.12;
    case ATOM_TYPE::SOL:
        return 0.05;
    case ATOM_TYPE::P:
        return 0.15;
    case ATOM_TYPE::NONE:
        return 1;
    default:
        return 1;
    }
}









__global__ void loadCompoundatomsKernel(Box* box, RenderAtom* atoms, const int step) {

    int local_id = threadIdx.x;
    int compound_id = blockIdx.x;
    int global_id = threadIdx.x + blockIdx.x * blockDim.x;

    Compound* compound = &box->compounds[compound_id];
    
    if (local_id < compound->n_particles) {
        auto coordarray_ptr = CoordArrayQueueHelpers::getCoordarrayRef(box->coordarray_circular_queue, step, compound_id);

        RenderAtom atom{};
        //atom.pos = LIMAPOSITIONSYSTEM::getGlobalPosition(*coordarray_ptr);
        atom.pos = LIMAPOSITIONSYSTEM::getAbsolutePositionNM(coordarray_ptr->origo, coordarray_ptr->rel_positions[local_id]);


        //atoms[global_id].pos.print('A');
        atom.mass = SOLVENT_MASS;                                                         // TEMP
        //atoms[global_id].atom_type = RAS_getTypeFromIndex(box->compounds[compound_id].atom_types[local_id]);
        atom.atom_type = RAS_getTypeFromIndex(compound->atom_color_types[local_id]);

        atom.color = getColor(atom.atom_type);

        atoms[global_id] = atom;
    }
    else {
        atoms[global_id].atom_type = ATOM_TYPE::NONE;
    }
}

__global__ void loadSolventatomsKernel(Box* box, RenderAtom* atoms, int offset, const int step) {
    SolventBlock* solventblock = CoordArrayQueueHelpers::getSolventBlockPtr(box->solventblockgrid_circular_queue, step, blockIdx.x);
    SolventBlock* solventblock_prev = CoordArrayQueueHelpers::getSolventBlockPtr(box->solventblockgrid_circular_queue, step == 0 ? 0 : step-1, blockIdx.x);

    if (threadIdx.x < solventblock->n_solvents) {
        //const SolventCoord coord{solventblock->origo, solventblock->rel_pos[threadIdx.x] };

		RenderAtom atom{};
		//atom.pos = coord.getAbsolutePositionLM();
        atom.pos = LIMAPOSITIONSYSTEM::getAbsolutePositionNM(solventblock->origo, solventblock->rel_pos[threadIdx.x]);
        EngineUtils::applyPBCNM(&atom.pos);   // TMP, dunno if i wanna do this.
		atom.mass = SOLVENT_MASS;
		atom.atom_type = SOL;

		// Debug
		//float velocity = (atom.pos - SolventCoord{ solventblock_prev->origo, solventblock_prev->rel_pos[threadIdx.x] }.getAbsolutePositionLM()).len();
        float velocity = 1.f;
        float point1nm = NANO_TO_LIMA * 0.1f;
		float color_scalar = velocity / point1nm * 255.f;
		uint8_t color_red = static_cast<uint8_t>(cuda::std::__clamp_to_integral<uint8_t, float>(color_scalar));
		atom.color = Int3(color_red, 0, 255 - color_red);
		//printf("vel %f, %d\n", velocity, color_red);

        // This part is for various debugging purposes
        //int query_id = 0;
        //if (solvent_id == query_id) {
        //    atoms[solvent_id + offset].atom_type = P;
        //}
        //const auto& nlist = box->solvent_neighborlists[solvent_id];
        //for (int i = 0; i < nlist.n_solvent_neighbors; i++) {
        //    if (nlist.neighborsolvent_ids[i] == query_id) {
        //        atoms[solvent_id + offset].atom_type = O;
        //    }
        //}
        int global_id = threadIdx.x + blockIdx.x * blockDim.x;
        atoms[global_id] = atom;
        //if (solvent_id != 440)
        //    atoms[solvent_id + offset].atom_type = NONE;
    }
}

__global__ void processAtomsKernel(RenderAtom* atoms, RenderBall* balls) { 
    const int index = threadIdx.x + blockIdx.x * RAS_THREADS_PER_BLOCK;
    
    RenderAtom atom = atoms[index];

    
    //atom.color = getColor(atom.atom_type);

    atom.radius = (getRadius(atom.atom_type)) / (1.f+atom.pos.y * 0.00000000001f);       // [nm]

    // Convert units to normalized units for OpenGL
    atom.radius = 0.25f * atom.radius;            // Yeah, i'm just eyeballing this..

    for (int dim = 0; dim < 3; dim++) {
        *atom.pos.placeAt(dim) = (atom.pos.at(dim) / BOX_LEN_NM - 0.5f) *1.8f;
    }
    
    const RenderBall ball(atom.pos, atom.radius, atom.color, atom.atom_type);
    balls[index] = ball;
}
