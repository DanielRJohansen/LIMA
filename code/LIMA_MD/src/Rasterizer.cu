#include "hip/hip_runtime.h"

#include "Rasterizer.cuh"
#include "Utilities.h"
#include "RenderUtilities.cuh"

using namespace RenderUtilities;

__global__ void loadCompoundatomsKernel(RenderAtom* atoms, const int step, const Float3* positions, const Compound* compounds, ColoringMethod coloringMethod, float boxLenNM);
__global__ void loadSolventatomsKernel(const Float3* positions, int n_compounds, int n_solvents, RenderAtom* atoms, float boxLenNM, int nCompoundParticles);

const bool drawSolvent = true;
const bool drawHydrogens = false;

void Rasterizer::initialize(const BoxParams& boxparams, const std::vector<Compound>& compounds) {
    hipMallocManaged(&positions_dev, sizeof(Float3) * boxparams.total_particles_upperbound);
    hipMallocManaged(&compounds_dev, sizeof(Compound) * boxparams.n_compounds);

    hipMemcpy(compounds_dev, compounds.data(), sizeof(Compound) * boxparams.n_compounds, hipMemcpyHostToDevice);

	isInitialized = true;
}

Rasterizer::~Rasterizer() {
    if (isInitialized) {
        hipFree(positions_dev);
        hipFree(compounds_dev);
    }
}

/// <summary>	/// Returns a pointer to a list of atoms on the device	/// </summary>
void Rasterizer::getAllAtoms(const Float3* positions, const std::vector<Compound>& compounds, const BoxParams& boxparams, int64_t step, ColoringMethod coloringMethod, RenderAtom* renderAtoms) {
    hipMemcpy(positions_dev, positions, sizeof(Float3) * boxparams.total_particles_upperbound, hipMemcpyHostToDevice);
    LIMA_UTILS::genericErrorCheck("Error before 1");

    if (boxparams.n_compounds > 0) {
        loadCompoundatomsKernel << <boxparams.n_compounds, MAX_COMPOUND_PARTICLES >> > (renderAtoms, step, positions_dev, compounds_dev, coloringMethod, static_cast<float>(boxparams.boxSize));
    }    LIMA_UTILS::genericErrorCheck("Error before 2");

    if (boxparams.n_solvents > 0) {
        loadSolventatomsKernel << < boxparams.n_solvents / 128 + 1, 128 >> > (positions_dev, boxparams.n_compounds, boxparams.n_solvents, renderAtoms, static_cast<float>(boxparams.boxSize), boxparams.total_compound_particles);   // TODO: This nsol/128 is wrong, if its not a multiple of 128
    }    LIMA_UTILS::genericErrorCheck("Error before 3");

}

void Rasterizer::render(const Float3* positions, const std::vector<Compound>& compounds, const BoxParams& boxparams, 
    int64_t step, Float3 camera_normal, ColoringMethod coloringMethod, RenderAtom* renderAtoms) {

    if (!isInitialized) { initialize(boxparams, compounds); }

    LIMA_UTILS::genericErrorCheck("Error before renderer");
	getAllAtoms(positions, compounds, boxparams, step, coloringMethod, renderAtoms);

    LIMA_UTILS::genericErrorCheck("Error after renderer\n");
}








/// <summary>
/// 
/// </summary>
/// <param name="box"></param>
/// <param name="atoms"></param>
/// <param name="step"></param>
/// <param name="positions">Absolute positions in nm of all particles (compounds first then solvents)</param>
/// <returns></returns>
__global__ void loadCompoundatomsKernel(RenderAtom* atoms, const int step, const Float3* positions, const Compound* compounds, ColoringMethod coloringMethod, float boxLenNM) {

    const int local_id = threadIdx.x;
    const int compound_id = blockIdx.x;
    const int global_id = threadIdx.x + blockIdx.x * blockDim.x;

    const int outputIndex = compounds[compound_id].absoluteIndexOfFirstParticle + threadIdx.x;

    const Compound* compound = &compounds[compound_id];
    
    if (local_id < compound->n_particles) {

        RenderAtom atom{};
        const Float3 positionNormalized = positions[global_id] / boxLenNM - 0.5f;// normalize from -0.5->0.5
        atom.position = float4{ positionNormalized.x, positionNormalized.y, positionNormalized.z, 0.f };        

        ATOM_TYPE atomType = RAS_getTypeFromAtomletter(compound->atomLetters[local_id]);

        atom.position.w = getRadius(atomType) / boxLenNM;
        if (coloringMethod == ColoringMethod::Atomname)
            atom.color = getColor(atomType);
        else if (coloringMethod == ColoringMethod::Charge) {
            const float chargeNormalized = (static_cast<float>(compound->atom_charges[local_id]) + elementaryChargeToKiloCoulombPerMole)  / (elementaryChargeToKiloCoulombPerMole *2.f);
            atom.color = float4{ chargeNormalized, 0.f, (1.f - chargeNormalized), 1.f };
        }

        if (atomType == ATOM_TYPE::H && !drawHydrogens) {
			atom.position.w = 0.f;
		}

        atoms[outputIndex] = atom;
    }
}

__global__ void loadSolventatomsKernel(const Float3* positions, int n_compounds, int n_solvents, RenderAtom* atoms, float boxLenNM, int nCompoundParticles)
{
    const int solvent_index = blockIdx.x * blockDim.x + threadIdx.x;
    const int particle_index = n_compounds * MAX_COMPOUND_PARTICLES + solvent_index;
    const int outputIndex = nCompoundParticles + solvent_index;

    if (solvent_index < n_solvents) {

		RenderAtom atom{};
        const Float3 positionNormalized = positions[particle_index] / boxLenNM - 0.5f;// normalize from -0.5->0.5
        atom.position = float4{ positionNormalized.x, positionNormalized.y, positionNormalized.z, 0.f };
        atom.position.w = getRadius(ATOM_TYPE::SOL) / boxLenNM;

        atom.color = float4{ 0,0,1, drawSolvent };

        atoms[outputIndex] = atom;
    }
}
